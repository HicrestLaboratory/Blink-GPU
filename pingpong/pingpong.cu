#include "hip/hip_runtime.h"
// Copyright 2009-2018 Sandia Corporation. Under the terms
// of Contract DE-NA0003525 with Sandia Corporation, the U.S.
// Government retains certain rights in this software.
//
// Copyright (c) 2009-2018, Sandia Corporation
// All rights reserved.
//
// Portions are copyright of other developers:
// See the file CONTRIBUTORS.TXT in the top level directory
// the distribution for more information.
//
// This file is part of the SST software package. For license
// information, see the LICENSE file in the top level directory of the
// distribution.

#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#include "../include/hip/hip_runtime_api.h"
#include "../include/experiment_utils.h"
// #include <../lib/llvm-13/lib/clang/13.0.1/include/stddef.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <nccl.h>

// #define DEBUG 1
#define NCCL
#include "../include/debug_utils.h"

#define CUDA
#define BLK_SIZE 256
#define GRD_SIZE 4

#define PINGPONG_REPEATS 1000
#define PINGPONG_MSG_SIZE 1024

#ifdef CUDA

static int stringCmp( const void *a, const void *b) {
     return strcmp((const char*)a,(const char*)b);

}

int  assignDeviceToProcess() {
      char     host_name[MPI_MAX_PROCESSOR_NAME];
      char (*host_names)[MPI_MAX_PROCESSOR_NAME];
      MPI_Comm nodeComm;

      int myrank;
      int gpu_per_node;
      int n, namelen, color, rank, nprocs;
      size_t bytes;

      MPI_Comm_rank(MPI_COMM_WORLD, &rank);
      MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
      MPI_Get_processor_name(host_name,&namelen);

      bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
      host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

      strcpy(host_names[rank], host_name);

      for (n=0; n<nprocs; n++)
      {
       MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR,n, MPI_COMM_WORLD);
      }


      qsort(host_names, nprocs, sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

      color = 0;

      for (n=0; n<nprocs; n++)
      {
        if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
        if(strcmp(host_name, host_names[n]) == 0) break;
      }

      MPI_Comm_split(MPI_COMM_WORLD, color, 0, &nodeComm);

      MPI_Comm_rank(nodeComm, &myrank);
      MPI_Comm_size(nodeComm, &gpu_per_node);

      return myrank;
}

__global__
void init_kernel(int n, char *input, int scale) {

  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  for (int i=0; i<n; i++) {
    int val_coord = tid * scale;
    if (tid < n)
        input[tid] = (char)val_coord;
  }
}

unsigned long long int check_result(int n, int myid, char* dev_recvBuffer, int recvid) {

  unsigned long long int error = 0ULL;
  char* checkBuffer = (char*)malloc(sizeof(char*) * n);

  checkCudaErrors( hipMemcpy(checkBuffer, dev_recvBuffer, n*sizeof(char), hipMemcpyDeviceToHost) );
  checkCudaErrors( hipDeviceSynchronize() );

  for (int i=0; i<n; i++)
    error += (unsigned long long int)(recvid*i - checkBuffer[i]);

  free(checkBuffer);

  return(error);
}

#endif

int main(int argc, char* argv[]) {

  MPI_Init(&argc, &argv);

  int me = -1;
  int world = -1;
  double timeTaken = 0.0, timeTakenCUDA = 0.0, TotalTimeTaken = 0.0;
  unsigned long long int interror = 0ULL;

  MPI_Comm_rank(MPI_COMM_WORLD, &me);
  MPI_Comm_size(MPI_COMM_WORLD, &world);

  DBG_CHECK(1)

  int msgSize = PINGPONG_MSG_SIZE;
  int repeats = PINGPONG_REPEATS;

  if (argc > 1) {
    msgSize = atoi(argv[1]);
  }

  if (argc > 2) {
    repeats = atoi(argv[2]);
  }

  if (0 == me) {
    printf("# MPI PingPong Pattern\n");
    printf("# Info:\n");
    printf("# - Total Ranks:     %10d\n", world);
    printf("# - Message Size:    %10d Bytes\n", msgSize);
    printf("# - Repeats:         %10d\n", repeats);
  }

  if (world < 2) {
    printf("No MPI is run because there are not 2 or more processors.\n");
    return 1;
  }

  MPI_Barrier(MPI_COMM_WORLD);

#ifdef CUDA
  int dev, deviceCount = 0;

  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (me == 0) {
    printf("#\n");
    printf("# Number of GPUs: %d\n", deviceCount);
  }

  dev = assignDeviceToProcess();
  hipSetDevice(dev);

  DBG_CHECK(1)

  MPI_Barrier(MPI_COMM_WORLD);
  for (int i=0; i<world; i++) {
    if (me == i)
      printf("#\tMPI process %d has device %d\n", me, dev);
    MPI_Barrier(MPI_COMM_WORLD);
  }
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  char* sendBuffer   = (char*)malloc(sizeof(char*) * msgSize);
  char* recvBuffer   = (char*)malloc(sizeof(char*) * msgSize);

  for (int i = 0; i < msgSize; ++i) {
    sendBuffer[i]  = 0;
    recvBuffer[i]  = 0;
  }

  MPI_Status status;

  DBG_CHECK(1)

#ifdef CUDA
  char *dev_sendBuffer, *dev_recvBuffer;
  checkCudaErrors( hipMalloc(&dev_sendBuffer, msgSize*sizeof(char)) );
  checkCudaErrors( hipMalloc(&dev_recvBuffer, msgSize*sizeof(char)) );
  checkCudaErrors( hipMemset(dev_sendBuffer, 0, msgSize*sizeof(char)) );
  checkCudaErrors( hipMemset(dev_recvBuffer, 0, msgSize*sizeof(char)) );
#endif

//   struct timeval startMPI, startCUDA;
//   struct timeval endMPI, endCUDA;
  TIMER_DEF(0);
  TIMER_DEF(1);

  DBG_CHECK(1)

#ifdef CUDA
  dim3 block_size(BLK_SIZE, 1, 1);
  dim3 grid_size(GRD_SIZE, 1, 1);
  printf("block_size = %d, grid_size = %d, elements per thread = %f\n", block_size.x, grid_size.x, (float)msgSize/(block_size.x*grid_size.x));
  init_kernel<<<grid_size, block_size>>>(msgSize, dev_sendBuffer, me);
  checkCudaErrors( hipDeviceSynchronize() );
#else
  for (int i = 0; i < msgSize; ++i)
    sendBuffer[i]  = i;
#endif


  DBG_CHECK(1)

  INIT_EXPS

  if (0 == me) {
    printf("# Beginning benchmarking...\n");
    printf("# ------------- Start MPI+memcpy -------------\n");
  }
  SET_EXPERIMENT(0, "MPI+memcpy")
  MPI_Barrier(MPI_COMM_WORLD);

  if (me < 2) {


    for (int i = 0; i < repeats; ++i) {

#ifdef CUDA
      TIMER_START(1);
      checkCudaErrors( hipMemcpy(sendBuffer, dev_sendBuffer, msgSize*sizeof(char), hipMemcpyDeviceToHost) );
      checkCudaErrors( hipDeviceSynchronize() );
      TIMER_STOP(1);

      timeTakenCUDA += TIMER_ELAPSED(1);
#endif

      TIMER_START(0);
      if (0 == me) {
        MPI_Send(sendBuffer, msgSize, MPI_CHAR, 1, 0, MPI_COMM_WORLD);
        MPI_Recv(recvBuffer, msgSize, MPI_CHAR, 1, 1, MPI_COMM_WORLD, &status);
      } else {
        MPI_Recv(recvBuffer, msgSize, MPI_CHAR, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Send(sendBuffer, msgSize, MPI_CHAR, 0, 1, MPI_COMM_WORLD);
      }
      TIMER_STOP(0);

      timeTaken += TIMER_ELAPSED(0);

#ifdef CUDA
      TIMER_START(1);
      checkCudaErrors( hipMemcpy(dev_recvBuffer, recvBuffer, msgSize*sizeof(char), hipMemcpyHostToDevice) );
      checkCudaErrors( hipDeviceSynchronize() );
      TIMER_STOP(1);

      timeTakenCUDA += TIMER_ELAPSED(1);
      interror = check_result( msgSize, me, dev_recvBuffer, ((me==0)?1:0) );
      ADD_INTERROR_EXPERIMENT(0, interror);

      checkCudaErrors( hipMemset(dev_recvBuffer, 0, msgSize*sizeof(char)) );
#endif

    }

    DBG_CHECK(1)

    free(sendBuffer);
    free(recvBuffer);

    if (0 == me) {
      printf("# Statistics:\n");

      const double bytesXchng = ((double)msgSize) * 2.0 * ((double)repeats);
      const double MbytesXchng = bytesXchng / (1024.0 * 1024.0);
      const double msgsXchng = ((double)repeats) * 2.0;
      const double KMsgsXchng = msgsXchng / 1000.0;

      printf("#%10s %9s %11s %17s %14s %16s %14s\n", "Type", "MsgSize", "Time", "KMsgs",
             "MB", "KMsg/S", "MB/S");
      printf("%10s  %9.0f %11.4f %17.5f %14.4f %16.4f %14.4f\n", "MPI", (double)msgSize,
             timeTaken, KMsgsXchng, MbytesXchng, KMsgsXchng / timeTaken,
             MbytesXchng / timeTaken);
#ifdef CUDA
      printf("%10s  %9.0f %11.4f %17.5f %14.4f %16.4f %14.4f\n", "memcpy", (double)msgSize,
             timeTakenCUDA, KMsgsXchng, MbytesXchng, KMsgsXchng / timeTakenCUDA,
             MbytesXchng / timeTakenCUDA);
      TotalTimeTaken = timeTaken + timeTakenCUDA;
      printf("%10s  %9.0f %11.4f %17.5f %14.4f %16.4f %14.4f\n", "MPI+memcpy", (double)msgSize,
             TotalTimeTaken, KMsgsXchng, MbytesXchng, KMsgsXchng / TotalTimeTaken,
             MbytesXchng / TotalTimeTaken);
#endif
      ADD_TIME_EXPERIMENT(0, TotalTimeTaken)
    }
  }

  if (0 == me) printf("# ---------------- Start NCCL ----------------\n");
  SET_EXPERIMENT(1, "NCCL")
  MPI_Barrier(MPI_COMM_WORLD);
  DBG_CHECK(1)

#ifdef NCCL
  interror = 0ULL;
  timeTaken = 0.0;
  checkCudaErrors( hipMemset(dev_sendBuffer, 0, msgSize*sizeof(char)) );
  checkCudaErrors( hipMemset(dev_recvBuffer, 0, msgSize*sizeof(char)) );
  init_kernel<<<grid_size, block_size>>>(msgSize, dev_sendBuffer, me);
  checkCudaErrors( hipDeviceSynchronize() );
  DBG_CHECK(1)

  ncclUniqueId Id;
  ncclComm_t NCCL_COMM_WORLD;
  if (me == 0) { NCCLCHECK( ncclGetUniqueId(&Id) ); }
  MPI_Bcast(&Id, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
  NCCLCHECK( ncclCommInitRank(&NCCL_COMM_WORLD, world, Id, me) );
  DBG_CHECK(1)

  for (int i = 0; i < repeats; ++i) {
    TIMER_START(0);
    ncclGroupStart();
    DBG_CHECK(1)
    if (me == 0) {
      DBG_CHECK(1)
      ncclSend(dev_sendBuffer, msgSize, ncclChar, 1, NCCL_COMM_WORLD, NULL);
      ncclRecv(dev_recvBuffer, msgSize, ncclChar, 1, NCCL_COMM_WORLD, NULL);
      DBG_CHECK(1)
    } else if (me == 1) {
      DBG_CHECK(1)
      ncclSend(dev_sendBuffer, msgSize, ncclChar, 0, NCCL_COMM_WORLD, NULL);
      ncclRecv(dev_recvBuffer, msgSize, ncclChar, 0, NCCL_COMM_WORLD, NULL);
      DBG_CHECK(1)
    }
    DBG_CHECK(1)
    ncclGroupEnd();
    TIMER_STOP(0);
    DBG_CHECK(1)

    timeTaken = TIMER_ELAPSED(0);
    interror = check_result( msgSize, me, dev_recvBuffer, ((me==0)?1:0) );
    ADD_INTERROR_EXPERIMENT(1, interror);
    ADD_TIME_EXPERIMENT(1, timeTaken);
  }
  DBG_CHECK(1)
#endif

  MPI_Barrier(MPI_COMM_WORLD);
  if (0 == me) printf("# --------------------------------------------\n");
  if (me == 0) PRINT_EXPARIMENT_STATS

  MPI_Finalize();
  return 0;
}
