#include "hip/hip_runtime.h"
// Copyright 2009-2018 Sandia Corporation. Under the terms
// of Contract DE-NA0003525 with Sandia Corporation, the U.S.
// Government retains certain rights in this software.
//
// Copyright (c) 2009-2018, Sandia Corporation
// All rights reserved.
//
// Portions are copyright of other developers:
// See the file CONTRIBUTORS.TXT in the top level directory
// the distribution for more information.
//
// This file is part of the SST software package. For license
// information, see the LICENSE file in the top level directory of the
// distribution.

#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define MPI
#define CUDA
#define NVLINK

#ifdef CUDA
#include "../../include/hip/hip_runtime_api.h"
#include "../../include/experiment_utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif

#ifdef NCCL
#include <nccl.h>
#endif

#ifdef GPUDIRECT

#ifdef __cplusplus
extern "C" {
#endif

#include <nv-p2p.h>

  #ifdef __cplusplus
}
#endif

#include <builtin_types.h>
// for boundary alignment requirement
#define GPU_BOUND_SHIFT   16
#define GPU_BOUND_SIZE    ((uint64_t)1 << GPU_BOUND_SHIFT)
#define GPU_BOUND_OFFSET  (GPU_BOUND_SIZE-1)
#define GPU_BOUND_MASK    (~GPU_BOUND_OFFSET)

#endif

// #define DEBUG 3
#include "../../include/debug_utils.h"

#define BLK_SIZE 256
#define GRD_SIZE 4

#define PINGPONG_REPEATS 1000
#define PINGPONG_MSG_SIZE 1024

#ifdef CUDA

static int stringCmp( const void *a, const void *b) {
     return strcmp((const char*)a,(const char*)b);

}

int  assignDeviceToProcess(MPI_Comm *nodeComm, int *nnodes, int *mynodeid)
{
#ifdef MPI
      char     host_name[MPI_MAX_PROCESSOR_NAME];
      char (*host_names)[MPI_MAX_PROCESSOR_NAME];

#else
      char     host_name[20];
#endif
      int myrank;
      int gpu_per_node;
      int n, namelen, color, rank, nprocs;
      size_t bytes;
/*
      if (chkseGPU()<1 && 0) {
        fprintf(stderr, "Invalid GPU Serial number\n");
	exit(EXIT_FAILURE);
      }
*/

#ifdef MPI
      MPI_Comm_rank(MPI_COMM_WORLD, &rank);
      MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
      MPI_Get_processor_name(host_name,&namelen);

      bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
      host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

      strcpy(host_names[rank], host_name);

      for (n=0; n<nprocs; n++)
      {
       MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR,n, MPI_COMM_WORLD);
      }


      qsort(host_names, nprocs, sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

      color = 0;

      for (n=0; n<nprocs; n++)
      {
        if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
        if(strcmp(host_name, host_names[n]) == 0) break;
      }

      MPI_Comm_split(MPI_COMM_WORLD, color, 0, nodeComm);

      MPI_Comm_rank(*nodeComm, &myrank);
      MPI_Comm_size(*nodeComm, &gpu_per_node);

      MPI_Allreduce(&color, nnodes, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
      (*mynodeid) = color;
      (*nnodes) ++;

#else
     //*myrank = 0;
     return 0;
#endif

//      printf ("Assigning device %d  to process on node %s rank %d\n",*myrank,  host_name, rank );
      /* Assign device to MPI process, initialize BLAS and probe device properties */
      //hipSetDevice(*myrank);
      return myrank;
}

__global__
void init_kernel(int n, char *input, int scale) {

  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  for (int i=0; i<n; i++) {
    int val_coord = tid * scale;
    if (tid < n)
        input[tid] = (char)val_coord;
  }
}

unsigned long long int check_result(int n, int myid, char* dev_recvBuffer, int recvid) {

  unsigned long long int error = 0ULL;
  char* checkBuffer = (char*)malloc(sizeof(char*) * n);

  checkCudaErrors( hipMemcpy(checkBuffer, dev_recvBuffer, n*sizeof(char), hipMemcpyDeviceToHost) );
  checkCudaErrors( hipDeviceSynchronize() );

  for (int i=0; i<n; i++)
    error += (unsigned long long int)((recvid+1)*i - checkBuffer[i]);

  free(checkBuffer);

  return(error);
}

#endif

int main(int argc, char* argv[]) {

  MPI_Init(&argc, &argv);

  int me = -1, mynode = -1;
  int world = -1, nnodes = -1;
  double timeTaken = 0.0, timeTakenCUDA = 0.0, TotalTimeTaken = 0.0;
  unsigned long long int interror = 0ULL;

  MPI_Comm_rank(MPI_COMM_WORLD, &me);
  MPI_Comm_size(MPI_COMM_WORLD, &world);

  DBG_CHECK(1)

  int msgSize = PINGPONG_MSG_SIZE;
  int repeats = PINGPONG_REPEATS;

  if (argc > 1) {
    msgSize = atoi(argv[1]);
  }

  if (argc > 2) {
    repeats = atoi(argv[2]);
  }

  if (0 == me) {
    printf("# MPI PingPong Pattern\n");
    printf("# Info:\n");
    printf("# - Total Ranks:     %10d\n", world);
    printf("# - Message Size:    %10d Bytes\n", msgSize);
    printf("# - Repeats:         %10d\n", repeats);
  }

  if (world < 2) {
    printf("No MPI is run because there are not 2 or more processors.\n");
    return 1;
  }

  MPI_Barrier(MPI_COMM_WORLD);

#ifdef CUDA
  int dev, deviceCount = 0;

  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (me == 0) {
    printf("#\n");
    printf("# Number of GPUs: %d\n", deviceCount);
  }

  MPI_Comm nodeComm;
  dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
  hipSetDevice(dev);

  int mynodeid = -1, mynodesize = -1;
  MPI_Comm_rank(nodeComm, &mynodeid);
  MPI_Comm_size(nodeComm, &mynodesize);

  MPI_ALL_PRINT( fprintf(fp, "mydev is %d, mynode is %d, nnodes are %d, mynodeid is %d and mynodesize is %d\n", dev, mynode, nnodes, mynodeid, mynodesize); )
//   MPI_Barrier(MPI_COMM_WORLD);
//   exit(42);

  DBG_CHECK(1)

  MPI_Barrier(MPI_COMM_WORLD);
  for (int i=0; i<world; i++) {
    if (me == i)
      printf("#\tMPI process %d has device %d\n", me, dev);
    MPI_Barrier(MPI_COMM_WORLD);
  }
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  char* sendBuffer   = (char*)malloc(sizeof(char*) * msgSize);
  char* recvBuffer   = (char*)malloc(sizeof(char*) * msgSize);

  for (int i = 0; i < msgSize; ++i) {
    sendBuffer[i]  = 0;
    recvBuffer[i]  = 0;
  }

  MPI_Status status;

  DBG_CHECK(1)

#ifdef CUDA
  char *dev_sendBuffer, *dev_recvBuffer;
  checkCudaErrors( hipMalloc(&dev_sendBuffer, msgSize*sizeof(char)) );
  checkCudaErrors( hipMalloc(&dev_recvBuffer, msgSize*sizeof(char)) );
  checkCudaErrors( hipMemset(dev_sendBuffer, 0, msgSize*sizeof(char)) );
  checkCudaErrors( hipMemset(dev_recvBuffer, 0, msgSize*sizeof(char)) );
#endif

  TIMER_DEF(0);
  TIMER_DEF(1);

  DBG_CHECK(1)

#ifdef CUDA
  dim3 block_size(BLK_SIZE, 1, 1);
  dim3 grid_size(GRD_SIZE, 1, 1);
  printf("block_size = %d, grid_size = %d, elements per thread = %f\n", block_size.x, grid_size.x, (float)msgSize/(block_size.x*grid_size.x));
  init_kernel<<<grid_size, block_size>>>(msgSize, dev_sendBuffer, me+1);
  checkCudaErrors( hipDeviceSynchronize() );
#else
  for (int i = 0; i < msgSize; ++i)
    sendBuffer[i]  = i;
#endif


  DBG_CHECK(1)

  INIT_EXPS


  fflush(stdout);
  MPI_Barrier(MPI_COMM_WORLD);
  if (0 == me) printf("# ----------------- NV LINK ------------------\n");
  SET_EXPERIMENT(3, "NV LINK")
  DBG_CHECK(4)
  interror = 0ULL;
  timeTaken = 0.0;
  timeTakenCUDA = 0.0;

#ifdef NVLINK
//   if (mynodeid == 0 || mynodeid == 1) {
    int canAccess = -1;
    if (mynodeid == 0) {
      hipDeviceEnablePeerAccess(1,0);
      checkCudaErrors( hipDeviceCanAccessPeer(&canAccess, mynodeid, 1) );
    } else if (mynodeid == 1) {
      hipDeviceEnablePeerAccess(0,0);
      checkCudaErrors( hipDeviceCanAccessPeer(&canAccess, mynodeid, 0) );
    }
    checkCudaErrors( hipDeviceSynchronize() );


    int *my_devpointer = NULL, *recv_devpointer = NULL, *peer_devpointer = NULL, host_sendflag, host_recvflag;
    hipIpcMemHandle_t *memHandles[mynodesize], *peerHandle = NULL;
    host_sendflag = (mynodeid+1)*10;
    host_recvflag = -1;

    MPI_ALL_PRINT( fprintf(fp, "canAccess = %d\nBEFORE: host_sendflag = %d, host_recvflag = %d\n", canAccess, host_sendflag, host_recvflag); )

    checkCudaErrors( hipMalloc(&my_devpointer, sizeof(int)) );
    checkCudaErrors( hipMalloc(&recv_devpointer, sizeof(int)) );
    checkCudaErrors( hipMemcpy(my_devpointer, &host_sendflag, sizeof(int), hipMemcpyHostToDevice) );

    for (int i=0; i<mynodesize; i++)
      memHandles[i] = NULL;

    MPI_ALL_PRINT(
      fprintf(fp, "BEFORE: my_devpointer = %p, recv_devpointer = %p, peerHandle = %p\nmemHandles: ", my_devpointer, recv_devpointer, peerHandle);
      for (int i=0; i<mynodesize; i++)
        fprintf(fp, "%p ", memHandles[i]);
      fprintf(fp, "\n");
    )


    hipIpcMemHandle_t memHandle;
    checkCudaErrors( hipIpcGetMemHandle ( &memHandle, my_devpointer ) );
    MPI_Allgather(&memHandle, sizeof(memHandle), MPI_BYTE, memHandles, sizeof(memHandle), MPI_BYTE, nodeComm);

    if (mynodeid == 0) {
      peerHandle = memHandles[1];
    } else if (mynodeid == 1) {
      peerHandle = memHandles[0];
    }
    MPI_Barrier(MPI_COMM_WORLD);

    MPI_ALL_PRINT(
      fprintf(fp, "AFTER: my_devpointer = %p, recv_devpointer = %p, peerHandle = %p\nmemHandles: ", my_devpointer, recv_devpointer, peerHandle);
      for (int i=0; i<mynodesize; i++)
        fprintf(fp, "%p ", memHandles[i]);
      fprintf(fp, "\n");
    )

    if (mynodeid == 0 || mynodeid == 1) {
      if (mynodeid == 0) {
//         checkCudaErrors( hipMemcpyPeer(recv_devpointer, 0, peer_pointer, 1, sizeof(int)) );
        checkCudaErrors( hipIpcOpenMemHandle ((void**)&peer_devpointer, memHandle, hipIpcMemLazyEnablePeerAccess) );
        checkCudaErrors( hipMemcpy(recv_devpointer, peer_devpointer, sizeof(int), hipMemcpyDeviceToDevice) );
      }else {
//         checkCudaErrors( hipMemcpyPeer(recv_devpointer, 1, peer_pointer, 0, sizeof(int)) );
        checkCudaErrors( hipIpcOpenMemHandle ((void**)&peer_devpointer, memHandle, hipIpcMemLazyEnablePeerAccess) );
        checkCudaErrors( hipMemcpy(recv_devpointer, peer_devpointer, sizeof(int), hipMemcpyDeviceToDevice) );
      }
      checkCudaErrors( hipDeviceSynchronize() );
      checkCudaErrors( hipMemcpy(&host_recvflag, recv_devpointer, sizeof(int), hipMemcpyDeviceToHost) );
    }

    checkCudaErrors( hipDeviceSynchronize() );
    MPI_ALL_PRINT( fprintf(fp, "canAccess = %d\nAFTER: host_sendflag = %d, host_recvflag = %d\n", canAccess, host_sendflag, host_recvflag); )
//   }
#else
  if (0 == me) printf("# the NVLINK macro is disabled\n");
#endif

  MPI_Barrier(MPI_COMM_WORLD);
  if (0 == me) printf("# --------------------------------------------\n");
  if (me == 0) PRINT_EXPARIMENT_STATS

  MPI_Finalize();
  return 0;
}
