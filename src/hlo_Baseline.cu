#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>

#if !defined(OPEN_MPI) || !OPEN_MPI
#error This source code uses an Open MPI-specific extension
#endif

/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"

#include <time.h>
#include <sys/time.h>
#include "../include/experiment_utils.h"

#define dtype u_int8_t
#define MPI_dtype MPI_CHAR

#define BUFF_CYCLE 24

#define cktype int32_t
#define MPI_cktype MPI_INT

#define WARM_UP 5

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
        exit(0);                                                                            \
    }                                                                                     \
}while(0)

#define MPI

static int stringCmp( const void *a, const void *b) {
     return strcmp((const char*)a,(const char*)b);

}

int  assignDeviceToProcess(MPI_Comm *nodeComm, int *nnodes, int *mynodeid)
{
#ifdef MPI
      char     host_name[MPI_MAX_PROCESSOR_NAME];
      char (*host_names)[MPI_MAX_PROCESSOR_NAME];

#else
      char     host_name[20];
#endif
      int myrank;
      int gpu_per_node;
      int n, namelen, color, rank, nprocs;
      size_t bytes;
/*
      if (chkseGPU()<1 && 0) {
        fprintf(stderr, "Invalid GPU Serial number\n");
	exit(EXIT_FAILURE);
      }
*/

#ifdef MPI
      MPI_Comm_rank(MPI_COMM_WORLD, &rank);
      MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
      MPI_Get_processor_name(host_name,&namelen);

      bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
      host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

      strcpy(host_names[rank], host_name);

      for (n=0; n<nprocs; n++)
      {
       MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR,n, MPI_COMM_WORLD);
      }


      qsort(host_names, nprocs, sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

      color = 0;

      for (n=0; n<nprocs; n++)
      {
        if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
        if(strcmp(host_name, host_names[n]) == 0) break;
      }

      MPI_Comm_split(MPI_COMM_WORLD, color, 0, nodeComm);

      MPI_Comm_rank(*nodeComm, &myrank);
      MPI_Comm_size(*nodeComm, &gpu_per_node);

      MPI_Allreduce(&color, nnodes, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
      (*mynodeid) = color;
      (*nnodes) ++;

#else
     //*myrank = 0;
     return 0;
#endif

//      printf ("Assigning device %d  to process on node %s rank %d\n",*myrank,  host_name, rank );
      /* Assign device to MPI process, initialize BLAS and probe device properties */
      //hipSetDevice(*myrank);
      return myrank;
}


// ---------------------------- For GPU reduction -----------------------------
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include "../include/debug_utils.h"

struct char2int
{
  __host__ __device__ cktype operator()(const dtype &x) const
  {
    return static_cast<cktype>(x);
  }
};

int gpu_host_reduce(dtype* input_vec, int len, cktype* out_scalar) {
  int result = thrust::transform_reduce(thrust::host,
                                        input_vec, input_vec + len,
                                        char2int(),
                                        0,
                                        thrust::plus<cktype>());

  *out_scalar = result;

  return 0;
}

int gpu_device_reduce(dtype* d_input_vec, int len, cktype* out_scalar) {
  cktype result = thrust::transform_reduce(thrust::device,
                                        d_input_vec, d_input_vec + len,
                                        char2int(),
                                        0,
                                        thrust::plus<cktype>());

  *out_scalar = result;

  return 0;
}

// ------------------------------- For Halo 3D --------------------------------

void get_position(const int rank, const int pex, const int pey, const int pez,
                  int* myX, int* myY, int* myZ) {
  const int plane = rank % (pex * pey);
  *myY = plane / pex;
  *myX = (plane % pex) != 0 ? (plane % pex) : 0;
  *myZ = rank / (pex * pey);
}

int convert_position_to_rank(const int pX, const int pY, const int pZ,
                             const int myX, const int myY, const int myZ) {
  // Check if we are out of bounds on the grid
  if ((myX < 0) || (myY < 0) || (myZ < 0) || (myX >= pX) || (myY >= pY) ||
      (myZ >= pZ)) {
    return -1;
  } else {
    return (myZ * (pX * pY)) + (myY * pX) + myX;
  }
}

void read_line_parameters (int argc, char *argv[], int myrank,
                           int *nx,  int *ny,  int *nz,
                           int *pex, int *pey, int *pez,
                           int *repeats, int *vars, long *sleep ) {

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-nx") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -nx without a value.\n");
            }

            exit(-1);
        }

        *nx = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-ny") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -ny without a value.\n");
            }

            exit(-1);
        }

        *ny = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-nz") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -nz without a value.\n");
            }

            exit(-1);
        }

        *nz = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pex") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pex without a value.\n");
            }

            exit(-1);
        }

        *pex = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pey") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pey without a value.\n");
            }

            exit(-1);
        }

        *pey = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pez") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pez without a value.\n");
            }

            exit(-1);
        }

        *pez = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-iterations") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -iterations without a value.\n");
            }

            exit(-1);
        }

        *repeats = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-vars") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -vars without a value.\n");
            }

            exit(-1);
        }

        *vars = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-sleep") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -sleep without a value.\n");
            }

            exit(-1);
        }

        *sleep = atol(argv[i + 1]);
        ++i;
        } else {
        if (0 == myrank) {
            fprintf(stderr, "Unknown option: %s\n", argv[i]);
        }

        exit(-1);
        }
    }
}

// HB --> Host_buffer, DB --> Device_buffer, DT --> data type, SZ --> size
#define INIT_HALO3D_BUFFER(HB, DB, DT, SZ) {                \
    int rank; MPI_Comm_rank(MPI_COMM_WORLD, &rank);         \
    HB = (DT*)malloc(sizeof(DT) * SZ);                      \
    cudaErrorCheck( hipMalloc(&DB, sizeof(DT) * SZ) );     \
    cudaErrorCheck( hipMemset(DB, 0, sizeof(DT) * SZ) );   \
    cudaErrorCheck( hipDeviceSynchronize() );              \
}

#define FREE_HALO3D_BUFFER(HB, DB) {                \
    cudaErrorCheck( hipFree(DB) );                 \
    free(HB);                                       \
}

#define BLK_SIZE 256
#define GRD_SIZE 4
#define TID_DIGITS 10000

__global__
void init_kernel(int n, dtype *input, int rank) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  if (tid < n)
      input[tid] = (dtype)(rank+1);

}

void halo3d_run_axes(int BufferSize, int UpFlag, int DownFlag,
                     dtype *host_UpSendBuffer,   dtype *dev_UpSendBuffer,   dtype *host_UpRecvBuffer,   dtype *dev_UpRecvBuffer,
                     dtype *host_DownSendBuffer, dtype *dev_DownSendBuffer, dtype *host_DownRecvBuffer, dtype *dev_DownRecvBuffer,
                     double *timeTakenCUDA, double *timeTakenMPI, int tag) {
    TIMER_DEF(0);

    int requestcount = 0;
    MPI_Status* status;
    status = (MPI_Status*)malloc(sizeof(MPI_Status) * 4);

    MPI_Request* requests;
    requests = (MPI_Request*)malloc(sizeof(MPI_Request) * 4);
    // =================================================================================================================

    // ---------------------------------------
    TIMER_START(0);
    if (UpFlag > -1) {
      cudaErrorCheck( hipMemcpy(host_UpSendBuffer, dev_UpSendBuffer, BufferSize*sizeof(dtype), hipMemcpyDeviceToHost) );
    }
    cudaErrorCheck( hipDeviceSynchronize() );

    if (DownFlag > -1) {
      cudaErrorCheck( hipMemcpy(host_DownSendBuffer, dev_DownSendBuffer, BufferSize*sizeof(dtype), hipMemcpyDeviceToHost) );
    }
    cudaErrorCheck( hipDeviceSynchronize() );

    TIMER_STOP(0);
    *timeTakenCUDA += TIMER_ELAPSED(0);
    MPI_Barrier(MPI_COMM_WORLD);
    // ---------------------------------------

    TIMER_START(0);
    if (UpFlag > -1) {
      MPI_Irecv(host_UpRecvBuffer, BufferSize, MPI_dtype, UpFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(host_UpSendBuffer, BufferSize, MPI_dtype, UpFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
    }

    if (DownFlag > -1) {
      MPI_Irecv(host_DownRecvBuffer, BufferSize, MPI_dtype, DownFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(host_DownSendBuffer, BufferSize, MPI_dtype, DownFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
    }

    MPI_Waitall(requestcount, requests, status);
    requestcount = 0;
    TIMER_STOP(0);
    *timeTakenMPI += TIMER_ELAPSED(0);

    // ---------------------------------------
    TIMER_START(0);
    if (UpFlag > -1) {
      cudaErrorCheck( hipMemcpy(dev_UpRecvBuffer, host_UpRecvBuffer, BufferSize*sizeof(dtype), hipMemcpyHostToDevice) );
    }

    if (DownFlag > -1) {
      cudaErrorCheck( hipMemcpy(dev_DownRecvBuffer, host_DownRecvBuffer, BufferSize*sizeof(dtype), hipMemcpyHostToDevice) );
    }

    cudaErrorCheck( hipDeviceSynchronize() );
    TIMER_STOP(0);
    *timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // =================================================================================================================
}

unsigned int check_recv_buffer (int my_rank, char axe,
                                int UpFlag, dtype *dev_UpBuffer,
                                int DownFlag, dtype *dev_DownBuffer,
                                int BufferSize) {

    cktype UpCheck = 0, DownCheck = 0;
    if (UpFlag>-1) gpu_device_reduce(dev_UpBuffer, BufferSize, &UpCheck);
    if (DownFlag>-1) gpu_device_reduce(dev_DownBuffer, BufferSize, &DownCheck);

    unsigned int result = 0U;
    if ( UpFlag>-1 && UpCheck != BufferSize*(UpFlag+1) ) result |= 1U;
    if ( DownFlag>-1 && DownCheck != BufferSize*(DownFlag+1) ) result |= 2U;
//     printf("[BufferSize=%d, myRank=%d, axe=%c] UpFlag = %d, UpCheck = %d, DownFlag = %d, DownCheck = %d --> %u\n", BufferSize, my_rank, axe, UpFlag, UpCheck, DownFlag, DownCheck, result);
    return(result);
}

// ----------------------------------------------------------------------------


int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */




    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(host_name, &namelen);
    MPI_Barrier(MPI_COMM_WORLD);

    printf("Size = %d, myrank = %d, host_name = %s\n", size, rank, host_name);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Status stat;

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );
//     cudaErrorCheck( hipSetDevice(rank % num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    hipSetDevice(dev);

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);

    int rank2 = size-1;
    MPI_Barrier(MPI_COMM_WORLD);

    /* -------------------------------------------------------------------------------------------
        Halo 3D Initialization
    --------------------------------------------------------------------------------------------*/

    // Define halo3D parameters
    int pex = size, pey = 1, pez = 1;
    int nx = 10, ny = 10, nz = 10;
    long sleep = 1000;
    int repeats = 100;
    int vars = 1;

    // Read input parameters
    read_line_parameters(argc, argv, rank, &nx, &ny, &nz, &pex, &pey, &pez, &repeats, &vars, &sleep);
    MPI_Barrier(MPI_COMM_WORLD);

    // Check for correct phisical initizlization
    if ((pex * pey * pez) != size) {
        if (0 == rank) {
        fprintf(stderr, "Error: rank grid does not equal number of ranks.\n");
        fprintf(stderr, "%7d x %7d x %7d != %7d\n", pex, pey, pez, size);
        }

        exit(-1);
    }
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    // Print data
    if (rank == 0) {
        printf("# MPI Nearest Neighbor Communication\n");
        printf("# Info:\n");
        printf("# Processor Grid:         %7d x %7d x %7d\n", pex, pey, pez);
        printf("# Data Grid (per rank):   %7d x %7d x %7d\n", nx, ny, nz);
        printf("# Iterations:             %7d\n", repeats);
        printf("# Variables:              %7d\n", vars);
        printf("# Sleep:                  %7ld\n", sleep);
    }
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    // Compute position and flags
    int posX, posY, posZ;
    get_position(rank, pex, pey, pez, &posX, &posY, &posZ);

    int xUp = convert_position_to_rank(pex, pey, pez, posX + 1, posY, posZ);
    int yUp = convert_position_to_rank(pex, pey, pez, posX, posY + 1, posZ);
    int zUp = convert_position_to_rank(pex, pey, pez, posX, posY, posZ + 1);
    int xDown = convert_position_to_rank(pex, pey, pez, posX - 1, posY, posZ);
    int yDown = convert_position_to_rank(pex, pey, pez, posX, posY - 1, posZ);
    int zDown = convert_position_to_rank(pex, pey, pez, posX, posY, posZ - 1);

    // Declare buffers and sizes variables
    size_t xSize, ySize, zSize;

    dtype     *xUpSendBuffer,     *xUpRecvBuffer,     *xDownSendBuffer,     *xDownRecvBuffer;
    dtype *dev_xUpSendBuffer, *dev_xUpRecvBuffer, *dev_xDownSendBuffer, *dev_xDownRecvBuffer;

    dtype     *yUpSendBuffer,     *yUpRecvBuffer,     *yDownSendBuffer,     *yDownRecvBuffer;
    dtype *dev_yUpSendBuffer, *dev_yUpRecvBuffer, *dev_yDownSendBuffer, *dev_yDownRecvBuffer;

    dtype     *zUpSendBuffer,     *zUpRecvBuffer,     *zDownSendBuffer,     *zDownRecvBuffer;
    dtype *dev_zUpSendBuffer, *dev_zUpRecvBuffer, *dev_zDownSendBuffer, *dev_zDownRecvBuffer;


     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    int loop_count = 50;
    double start_time, stop_time;
    double cuda_timer[3], mpi_timer[3];
    unsigned int halo_checks[BUFF_CYCLE];
    double inner_elapsed_time[BUFF_CYCLE][loop_count], elapsed_time[BUFF_CYCLE][loop_count];
    for(int j=0; j<BUFF_CYCLE; j++){

        // Define cycle sizes
        long int N = 1 << j;
        xSize = ny * nz * N;
        ySize = nx * nz * N;
        zSize = nx * ny * N;
        halo_checks[j] = 0U;

//         STR_COLL_DEF
//         STR_COLL_INIT

        // Alloc x axe
        INIT_HALO3D_BUFFER(xUpSendBuffer, dev_xUpSendBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xUpRecvBuffer, dev_xUpRecvBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xDownSendBuffer, dev_xDownSendBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xDownRecvBuffer, dev_xDownRecvBuffer, dtype, xSize)

        // Alloc y axe
        INIT_HALO3D_BUFFER(yUpSendBuffer, dev_yUpSendBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yUpRecvBuffer, dev_yUpRecvBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yDownSendBuffer, dev_yDownSendBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yDownRecvBuffer, dev_yDownRecvBuffer, dtype, ySize)

        // Alloc z axe
        INIT_HALO3D_BUFFER(zUpSendBuffer, dev_zUpSendBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zUpRecvBuffer, dev_zUpRecvBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zDownSendBuffer, dev_zDownSendBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zDownRecvBuffer, dev_zDownRecvBuffer, dtype, zSize)

//         MPI_ALL_PRINT(fprintf(fp, "%s", STR_COLL_GIVE);)
//         STR_COLL_FREE

        cudaErrorCheck( hipDeviceSynchronize() );
        MPI_Barrier(MPI_COMM_WORLD);
        fflush(stdout);

        // Init send buffers (Recv buffers stay initialized as 0)
        {
            dim3 block_size(BLK_SIZE, 1, 1);
            dim3 grid_size(GRD_SIZE, 1, 1);
            init_kernel<<<grid_size, block_size>>>(xSize, dev_xUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(ySize, dev_yUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(zSize, dev_zUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(xSize, dev_xDownSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(ySize, dev_yDownSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(zSize, dev_zDownSendBuffer, rank);
            cudaErrorCheck( hipDeviceSynchronize() );
            MPI_Barrier(MPI_COMM_WORLD);
        }

        if (rank == 0) {printf("%d#", j); fflush(stdout);}
        cudaErrorCheck( hipDeviceSynchronize() );
        MPI_Barrier(MPI_COMM_WORLD);
        fflush(stdout);

        /*

        Implemetantion goes here

        */

        for(int i=1-(WARM_UP); i<=loop_count; i++) {
            for (int k=0; k<3; k++) {cuda_timer[k] = 0.0; mpi_timer[k] = 0.0;}
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

            halo3d_run_axes(xSize, xUp, xDown,
                     xUpSendBuffer, dev_xUpSendBuffer, xUpRecvBuffer, dev_xUpRecvBuffer,
                     xDownSendBuffer, dev_xDownSendBuffer, xDownRecvBuffer, dev_xDownRecvBuffer,
                     &(cuda_timer[0]), &(mpi_timer[0]), 1000);

            halo3d_run_axes(ySize, yUp, yDown,
                     yUpSendBuffer, dev_yUpSendBuffer, yUpRecvBuffer, dev_yUpRecvBuffer,
                     yDownSendBuffer, dev_yDownSendBuffer, yDownRecvBuffer, dev_yDownRecvBuffer,
                     &(cuda_timer[1]), &(mpi_timer[1]), 2000);

            halo3d_run_axes(zSize, zUp, zDown,
                     zUpSendBuffer, dev_zUpSendBuffer, zUpRecvBuffer, dev_zUpRecvBuffer,
                     zDownSendBuffer, dev_zDownSendBuffer, zDownRecvBuffer, dev_zDownRecvBuffer,
                     &(cuda_timer[2]), &(mpi_timer[2]), 3000);

            stop_time = MPI_Wtime();
            if (i>0) inner_elapsed_time[j][i-1] = stop_time - start_time;

            if (rank == 0) {printf("%%"); fflush(stdout);}

            unsigned int xCheck = check_recv_buffer(rank, 'x', xUp, dev_xUpRecvBuffer, xDown, dev_xDownRecvBuffer, xSize);
            unsigned int yCheck = check_recv_buffer(rank, 'y', yUp, dev_yUpRecvBuffer, yDown, dev_yDownRecvBuffer, ySize);
            unsigned int zCheck = check_recv_buffer(rank, 'z', zUp, dev_zUpRecvBuffer, zDown, dev_zDownRecvBuffer, zSize);
            xCheck = xCheck << 4;
            yCheck = yCheck << 2;
            halo_checks[j] |= xCheck;
            halo_checks[j] |= yCheck;
            halo_checks[j] |= zCheck;
        }
        if (rank == 0) {printf("#\n"); fflush(stdout);}



//         int tag1 = 10;
//         int tag2 = 20;
//         gpu_device_reduce(d_B, N, &gpu_check);
//         if(rank == 0){
//             MPI_Send(&my_cpu_check,   1, MPI_cktype, rank2, tag1, MPI_COMM_WORLD);
//             MPI_Recv(&recv_cpu_check, 1, MPI_cktype, rank2, tag2, MPI_COMM_WORLD, &stat);
//         } else if(rank == rank2){
//             MPI_Recv(&recv_cpu_check, 1, MPI_cktype, 0, tag1, MPI_COMM_WORLD, &stat);
//             MPI_Send(&my_cpu_check,   1, MPI_cktype, 0, tag2, MPI_COMM_WORLD);
//         }
//
//         gpu_checks[j] = gpu_check;
//         cpu_checks[j] = recv_cpu_check;

        // Free x axe
        FREE_HALO3D_BUFFER(xUpSendBuffer, dev_xUpSendBuffer)
        FREE_HALO3D_BUFFER(xUpRecvBuffer, dev_xUpRecvBuffer)
        FREE_HALO3D_BUFFER(xDownSendBuffer, dev_xDownSendBuffer)
        FREE_HALO3D_BUFFER(xDownRecvBuffer, dev_xDownRecvBuffer)

        // Free y axe
        FREE_HALO3D_BUFFER(yUpSendBuffer, dev_yUpSendBuffer)
        FREE_HALO3D_BUFFER(yUpRecvBuffer, dev_yUpRecvBuffer)
        FREE_HALO3D_BUFFER(yDownSendBuffer, dev_yDownSendBuffer)
        FREE_HALO3D_BUFFER(yDownRecvBuffer, dev_yDownRecvBuffer)

        // Free z axe
        FREE_HALO3D_BUFFER(zUpSendBuffer, dev_zUpSendBuffer)
        FREE_HALO3D_BUFFER(zUpRecvBuffer, dev_zUpRecvBuffer)
        FREE_HALO3D_BUFFER(zDownSendBuffer, dev_zDownSendBuffer)
        FREE_HALO3D_BUFFER(zDownRecvBuffer, dev_zDownRecvBuffer)
    }

    MPI_Allreduce(inner_elapsed_time, elapsed_time, BUFF_CYCLE*loop_count, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    for(int j=0; j<BUFF_CYCLE; j++) {
        long int N = 1 << j;
        long int B_in_GB = 1 << 30;
        long int num_B = 0;
        if (xUp > -1) num_B += ny * nz * N;
        if (yUp > -1) num_B += nx * nz * N;
        if (zUp > -1) num_B += nx * ny * N;
        if (xDown > -1) num_B += ny * nz * N;
        if (yDown > -1) num_B += nx * nz * N;
        if (zDown > -1) num_B += nx * ny * N;
        double num_GB = (double)num_B / (double)B_in_GB;

        double avg_time_per_transfer[BUFF_CYCLE];
        for (int i=0; i<loop_count; i++) {
            avg_time_per_transfer[j] += elapsed_time[j][i];
            if(rank == 0) printf("\tTransfer size (B): %10li, Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f, Iteration %d\n", num_B, inner_elapsed_time[j][i], num_GB/inner_elapsed_time[j][i], i);
        }
        avg_time_per_transfer[j] /= ((double)loop_count);

        if(rank == 0) printf("[Average] Transfer size (B): %10li, Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer[j], num_GB/avg_time_per_transfer[j], halo_checks[j] );
        fflush(stdout);
    }

//     char s[10000000];
//     sprintf(s, "[%d] recv_cpu_check = %u", rank, cpu_checks[0]);
//     for (int i=0; i<BUFF_CYCLE; i++) {
//         sprintf(s+strlen(s), " %10d", cpu_checks[i]);
//     }
//     sprintf(s+strlen(s), " (for Error)\n");
//     printf("%s", s);
//     fflush(stdout);
//
//     sprintf(s, "[%d] gpu_checks = %u", rank, gpu_checks[0]);
//     for (int i=0; i<BUFF_CYCLE; i++) {
//         sprintf(s+strlen(s), " %10d", gpu_checks[i]);
//     }
//     sprintf(s+strlen(s), " (for Error)\n");
//     printf("%s", s);
//     fflush(stdout);

    MPI_Finalize();
    return(0);
}
