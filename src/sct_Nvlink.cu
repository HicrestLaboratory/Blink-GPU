#include <stdio.h>
#include "mpi.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <inttypes.h>

#define MPI

#include "nvToolsExt.h"

#include "../include/error.h"
#include "../include/type.h"
#include "../include/gpu_ops.h"
#include "../include/device_assignment.h"
#include "../include/prints.h"
#include "../include/communicators.h"
#include "../include/common.h"

#ifdef MPIX_CUDA_AWARE_SUPPORT
/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"
#endif

#define BUFF_CYCLE 31
#define LOOP_COUNT 50

#define WARM_UP 5

// ---------------------------------------
void PICO_enable_peer_access(int myrank, int deviceCount, int mydev) {
    // Pick all the devices that can access each other's memory for this test
    // Keep in mind that CUDA has minimal support for fork() without a
    // corresponding exec() in the child process, but in this case our
    // spawnProcess will always exec, so no need to worry.
    hipDeviceProp_t prop;
    int allPeers = 1, myIPC = 1, allIPC;
    cudaErrorCheck(hipGetDeviceProperties(&prop, mydev));

    int* canAccesPeer = (int*) malloc(sizeof(int)*deviceCount*deviceCount);
    for (int i = 0; i < deviceCount*deviceCount; i++) canAccesPeer[i] = 0;

    // CUDA IPC is only supported on devices with unified addressing
    if (!prop.unifiedAddressing) {
      myIPC = 0;
    } else {
    }
    // This sample requires two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      myIPC = 0;
    }

    MPI_Allreduce(&myIPC, &allIPC, 1, MPI_INT, MPI_MIN, MPI_COMM_WORLD);
    if (!allIPC) {
      exit(__LINE__);
    }

    if (myrank == 0) {
      for (int i = 0; i < deviceCount; i++) {
        for (int j = 0; j < deviceCount; j++) {
          if (j != i) {
            int canAccessPeerIJ, canAccessPeerJI;
            cudaErrorCheck( hipDeviceCanAccessPeer(&canAccessPeerJI, j, i) );
            cudaErrorCheck( hipDeviceCanAccessPeer(&canAccessPeerIJ, i, j) );

            canAccesPeer[i * deviceCount + j] = (canAccessPeerIJ) ? 1 : 0;
            canAccesPeer[j * deviceCount + i] = (canAccessPeerJI) ? 1 : 0;
            if (!canAccessPeerIJ || !canAccessPeerJI) allPeers = 0;
          } else {
            canAccesPeer[i * deviceCount + j] = -1;
          }
        }
      }
    }

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&allPeers, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(canAccesPeer, deviceCount*deviceCount, MPI_INT, 0, MPI_COMM_WORLD);

    if (allPeers) {
      // Enable peers here.  This isn't necessary for IPC, but it will
      // setup the peers for the device.  For systems that only allow 8
      // peers per GPU at a time, this acts to remove devices from CanAccessPeer
      for (int j = 0; j < deviceCount; j++) {
        if (j != mydev) {
          cudaErrorCheck(hipDeviceEnablePeerAccess(j, 0));
        }
      }
    }

    MPI_Barrier(MPI_COMM_WORLD);
}

void PICO_disable_peer_access(int deviceCount, int mydev){
    MPI_Barrier(MPI_COMM_WORLD);
    for (int j = 0; j < deviceCount; j++) {
      if (j != mydev) {
        cudaErrorCheck(hipDeviceDisablePeerAccess(j));
      }
    }
}

void read_line_parameters (int argc, char *argv[], int myrank,
                           int *flag_b, int *flag_l, int *flag_x, int *flag_p,
                           int *loop_count, int *buff_cycle, int *fix_buff_size, int *ncouples ) {

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-l") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                    fprintf(stderr, "Error: specified -l without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_l = 1;
            *loop_count = atoi(argv[i + 1]);
            if (*loop_count <= 0) {
                fprintf(stderr, "Error: loop_count must be a positive integer.\n");
                exit(__LINE__);
            }
            i++;
        } else if (strcmp(argv[i], "-b") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                    fprintf(stderr, "Error: specified -b without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_b = 1;
            *buff_cycle = atoi(argv[i + 1]);
            if (*buff_cycle <= 0) {
                fprintf(stderr, "Error: buff_cycle must be a positive integer.\n");
                exit(__LINE__);
            }
            i++;
        } else if (strcmp(argv[i], "-x") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                fprintf(stderr, "Error: specified -x without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_x = 1;
            *fix_buff_size = atoi(argv[i + 1]);
            if (*fix_buff_size < 0) {
                fprintf(stderr, "Error: fixed buff_size must be >= 0.\n");
                exit(__LINE__);
            }

            i++;
        } else if (strcmp(argv[i], "-p") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                fprintf(stderr, "Error: specified -p without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_p = 1;
            *ncouples = atoi(argv[i + 1]);
            if (*ncouples < 0) {
                fprintf(stderr, "Error: number of ping-pong couples must be >= 1.\n");
                exit(__LINE__);
            }

            i++;
        } else {
            if (0 == myrank) {
                fprintf(stderr, "Unknown option: %s\n", argv[i]);
            }

            exit(__LINE__);
        }
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char *argv[])
{

    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
        int nnodes, mynode; // tmp
    int size, rank, namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MY_MPI_INIT(size, rank, namelen, host_name)

    MPI_Status stat;

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );
    //     cudaErrorCheck( hipSetDevice(rank % num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    hipSetDevice(dev);

    // print device affiniy
#ifndef SKIPCPUAFFINITY
    if (0==rank) printf("List device affinity:\n");
    check_cpu_and_gpu_affinity(dev);
    if (0==rank) printf("List device affinity done.\n\n");
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);

    // Check that all the nodes has the same size
    int nodesize;
    if (nnodes > 1) {
        MPI_Allreduce(&mynodesize, &nodesize, sizeof(int), MPI_INT, MPI_MAX, MPI_COMM_WORLD);
        if (nodesize != mynodesize) {
            fprintf(stderr, "Error at node %d: mynodesize (%d) does not metch with nodesize (%d)\n", rank, mynodesize, nodesize);
            fflush(stderr);
            MPI_Abort(MPI_COMM_WORLD, __LINE__);
        } else {
            if (rank == 0) printf("All the nodes (%d) have the same size (%d)\n", nnodes, nodesize);
            fflush(stdout);
        }
    } else {
        nodesize = mynodesize;
    }
    MPI_Barrier(MPI_COMM_WORLD);

    /* -------------------------------------------------------------------------------------------
        CUDA AWARE CHECK
    --------------------------------------------------------------------------------------------*/

    cudaAwareCheck();

    /* -------------------------------------------------------------------------------------------
        Reading command line inputs
    --------------------------------------------------------------------------------------------*/

    int opt;
    int max_j;
    int flag_b = 0;
    int flag_l = 0;
    int flag_x = 0;
    int flag_p = 0;
    int loop_count = LOOP_COUNT;
    int buff_cycle = BUFF_CYCLE;
    int fix_buff_size = 0;
    int ncouples = 4;

    // Parse command-line options
    read_line_parameters(argc, argv, rank,
                         &flag_b, &flag_l, &flag_x, &flag_p,
                         &loop_count, &buff_cycle, &fix_buff_size, &ncouples);
    if(flag_x && fix_buff_size >= buff_cycle){buff_cycle = fix_buff_size + 1;} 
       
    // Print message based on the flags
    if (flag_p && rank == 0) printf("Flag p was set with argument: %d\n", ncouples);
    if (flag_b && rank == 0) printf("Flag b was set with argument: %d\n", buff_cycle);
    if (flag_l && rank == 0) printf("Flag l was set with argument: %d\n", loop_count);
    if (flag_x && rank == 0) printf("Flag x was set with argument: %d\n", fix_buff_size);


    printf("[%d] DBG check at line %d\n", rank, __LINE__); fflush(stdout);

    if (flag_p) {
        if (nnodes > 1) {
            if (nodesize < ncouples) {
                fprintf(stderr, "Error: mynode (%s) has less gpus (%d) then the required by -p flag (%d)\n", host_name, nodesize, ncouples);
                MPI_Abort(MPI_COMM_WORLD, __LINE__);
            }
        } else {
            if (ncouples > 1) {
                fprintf(stderr, "Error: Multi-Ping-Pong does not support the single node set-up\n");
                MPI_Abort(MPI_COMM_WORLD, __LINE__);
            }
        }
    }

    if(!flag_p){ncouples = nodesize;}

    max_j = (flag_x == 0) ? buff_cycle : (fix_buff_size + 1) ;
    if (rank == 0) printf("buff_cycle: %d loop_count: %d max_j: %d\n", buff_cycle, loop_count, max_j);
    if (flag_x > 0 && rank == 0) printf("fix_buff_size is set as %d\n", fix_buff_size);


    printf("[%d] DBG check at line %d\n", rank, __LINE__); fflush(stdout);

    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    /* -------------------------------------------------------------------------------------------
        MPI Initialize Peer-to-peer communicators
    --------------------------------------------------------------------------------------------*/

    int my_peer = -1;
    if ((mynode == 0 || mynode == nnodes-1) && mynodeid < ncouples)
        my_peer = (mynode == 0) ? (rank + (nnodes-1)*nodesize) : (rank - (nnodes-1)*nodesize);

    printf("[%d] DBG check at line %d\n", rank, __LINE__); fflush(stdout);

    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    PICO_enable_peer_access(mynodeid, nodesize, dev);


    printf("[%d] DBG check at line %d\n", rank, __LINE__); fflush(stdout);

    SZTYPE N;
    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }


    printf("[%d] DBG check at line %d\n", rank, __LINE__); fflush(stdout);

    MPI_Status IPCstat;
    dtype *peerBBuffers[ncouples], *peerAggBuffer;
    hipEvent_t event;
    hipIpcMemHandle_t sendBHandle, recvBHandle[ncouples], sendAggHandle, recvAggHandle;

    hipStream_t Streams[4];
    double start_time, stop_time;
    double *elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    double *inner_elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
//     if (ppCouples != MPI_COMM_NULL) {
        for(int j=fix_buff_size; j<max_j; j++){

            (j!=0) ? (N <<= 1) : (N = 1);

            for (int k=0; k<4; k++) {cudaErrorCheck(hipStreamCreate(&Streams[k]));}

            // Allocate memory for A on CPU
            dtype *d_Agg;
            dtype *A, *B;
            alloc_host_buffers(rank, &A, N, &B, N);

            // Initialize all elements of A to 0.0
            INIT_HOST_BUFFER(A, N, 1U * (rank+1))
            INIT_HOST_BUFFER(B, N, 0U)

            dtype *d_A, *d_B;
            alloc_device_buffers(A, &d_A, N, B, &d_B, N);

            if (mynodeid == 0) {
                cudaErrorCheck( hipMalloc(&d_Agg, N*ncouples*sizeof(dtype)) );
                cudaErrorCheck( hipMemset(d_Agg, 0U, N*ncouples*sizeof(dtype)) );
            }

            int tag1 = 10;
            int tag2 = 20;
            MPI_Request request[2*ncouples];

            /*

            Implemetantion goes here

            */

            if (rank == 0) {printf("%i#", j); fflush(stdout);}

            PUSH_RANGE("initializeIPC", 0)

            // Generate IPC MemHandle
            cudaErrorCheck( hipIpcGetMemHandle((hipIpcMemHandle_t*)&sendBHandle, d_B) );
            if (mynodeid==0) { cudaErrorCheck( hipIpcGetMemHandle((hipIpcMemHandle_t*)&sendAggHandle, d_Agg) ); }

            // Share IPC MemHandle
            MPI_Gather(&sendBHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, &recvBHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, MPI_COMM_WORLD);
            if (mynodeid == 0)
                MPI_Bcast(&sendAggHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, MPI_COMM_WORLD);
            else
                MPI_Bcast(&recvAggHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, MPI_COMM_WORLD);

            // Open MemHandles
            if (mynodeid == 0) {
                for (int i=0; i<ncouples; i++) {
                    if (i != 0) {
                        cudaErrorCheck( hipIpcOpenMemHandle((void**)&peerBBuffers[i], *(hipIpcMemHandle_t*)&recvBHandle[i], hipIpcMemLazyEnablePeerAccess) );
                    } else {
                        peerBBuffers[i] = d_B;
                    }
                }
                peerAggBuffer = d_Agg;
            } else {
                cudaErrorCheck( hipIpcOpenMemHandle((void**)&peerAggBuffer, *(hipIpcMemHandle_t*)&recvAggHandle, hipIpcMemLazyEnablePeerAccess) );
            }
            MPI_Barrier(MPI_COMM_WORLD);
            POP_RANGE

            for(int i=1-(WARM_UP); i<=loop_count; i++){
                MPI_Barrier(MPI_COMM_WORLD);
                start_time = MPI_Wtime();


                for (int k=0; k<2; k++) {

                    PUSH_RANGE("IPCgather", 1)
                    // Aggregate d_A buffers into mynodeid 0 agg buffer
                    if (mynode == k*(nnodes-1)) {
                        cudaErrorCheck( hipMemcpyAsync(peerAggBuffer + (mynodeid*N), d_A, sizeof(dtype)*N, hipMemcpyDeviceToDevice, Streams[mynodeid]) );
                        cudaErrorCheck( hipDeviceSynchronize() );
                    }
                    MPI_Barrier(MPI_COMM_WORLD);
                    POP_RANGE
/*
                    // Out-node communication
                    if (ppOutCouple != MPI_COMM_NULL) {
                        if (mynode == k*(nnodes-1)) {
                            MPI_Send(d_Agg, ncouples*N, MPI_dtype, my_peer, tag1, MPI_COMM_WORLD);
                        } else {
                            MPI_Recv(d_Agg, ncouples*N, MPI_dtype, my_peer, tag1, MPI_COMM_WORLD, &stat);
                        }
                        MPI_Barrier(ppOutCouple);
                    }
                    MPI_Barrier(ppAllNodeCouples);
*/

                    PUSH_RANGE("IPCscatter", 2)
                    // Scatter agg buffer into d_B buffers
                    if (mynode == (1-k)*(nnodes-1) && mynodeid == 0) {
                        for (int i=0; i<ncouples; i++) {
                            cudaErrorCheck( hipMemcpyAsync(peerBBuffers[i], d_Agg + (i*N), sizeof(dtype)*N, hipMemcpyDeviceToDevice, Streams[i]) );
                        }
                        cudaErrorCheck( hipDeviceSynchronize() );
                    }
                    MPI_Barrier(MPI_COMM_WORLD);
                    POP_RANGE
                }

                stop_time = MPI_Wtime();
                if (i>0) inner_elapsed_time[(j-fix_buff_size)*loop_count+i-1] = stop_time - start_time;

                if (rank == 0) {printf("%%"); fflush(stdout);}
            }
            if (rank == 0) {printf("#\n"); fflush(stdout);}

            fflush(stdout);
            if (mynodeid == 0) cudaErrorCheck( hipFree(d_Agg) );
            cudaErrorCheck( hipFree(d_A) );
            cudaErrorCheck( hipFree(d_B) );
            free(A);
            free(B);

            for (int k=0; k<4; k++) {cudaErrorCheck(hipStreamDestroy(Streams[k]));}
        }

        if (fix_buff_size<=30) {
            N = 1 << (fix_buff_size - 1);
        } else {
            N = 1 << 30;
            N <<= (fix_buff_size - 31);
        }

//         if(ppFirstSenders != MPI_COMM_NULL) {
            MPI_Allreduce(inner_elapsed_time, elapsed_time, buff_cycle*loop_count, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
//         }
        for(int j=fix_buff_size; j<max_j; j++) {
            (j!=0) ? (N <<= 1) : (N = 1);

            SZTYPE num_B, int_num_GB;
            double num_GB;

            num_B = sizeof(dtype)*N*ncouples;
            // TODO: maybe we can avoid if and just divide always by B_in_GB
            if (j < 31) {
                SZTYPE B_in_GB = 1 << 30;
                num_GB = (double)num_B / (double)B_in_GB;
            } else {
                SZTYPE M = 1 << (j - 30);            
                num_GB = sizeof(dtype)*M;
            }

            double avg_time_per_transfer = 0.0;
            for (int i=0; i<loop_count; i++) {
                elapsed_time[(j-fix_buff_size)*loop_count+i] /= 2.0;
                avg_time_per_transfer += elapsed_time[(j-fix_buff_size)*loop_count+i];
                if(rank == 0) printf("\tTransfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f, Iteration %d\n", num_B, elapsed_time[(j-fix_buff_size)*loop_count+i], num_GB/elapsed_time[(j-fix_buff_size)*loop_count+i], i);
            }
            avg_time_per_transfer /= (double)loop_count;

            if(rank == 0) printf("[Average] Transfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, -1 );
            fflush(stdout);
        }
//     }
    free(elapsed_time);
    free(inner_elapsed_time);
    MPI_Finalize();
    return(0);
}
