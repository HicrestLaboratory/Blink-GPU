#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>
#include <unistd.h>
#include <inttypes.h>

#define MPI

#include "../include/error.h"
#include "../include/type.h"
#include "../include/gpu_ops.h"
#include "../include/device_assignment.h"
#include "../include/prints.h"

#ifdef MPIX_CUDA_AWARE_SUPPORT
/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"
#endif

#include <time.h>
#include <sys/time.h>
#include "../include/experiment_utils.h"

#define BUFF_CYCLE 24
#define LOOP_COUNT 50

#define WARM_UP 5

#include "../include/debug_utils.h"

// ------------------------------- For Halo 3D --------------------------------

void get_position(const int rank, const int pex, const int pey, const int pez,
                  int* myX, int* myY, int* myZ) {
  const int plane = rank % (pex * pey);
  *myY = plane / pex;
  *myX = (plane % pex) != 0 ? (plane % pex) : 0;
  *myZ = rank / (pex * pey);
}

int convert_position_to_rank(const int pX, const int pY, const int pZ,
                             const int myX, const int myY, const int myZ) {
  // Check if we are out of bounds on the grid
  if ((myX < 0) || (myY < 0) || (myZ < 0) || (myX >= pX) || (myY >= pY) ||
      (myZ >= pZ)) {
    return -1;
  } else {
    return (myZ * (pX * pY)) + (myY * pX) + myX;
  }
}

void read_line_parameters (int argc, char *argv[], int myrank,
                           int *nx,  int *ny,  int *nz,
                           int *pex, int *pey, int *pez,
                           int *repeats, int *vars, long *sleep,
                           int *flag_b, int *flag_l, int *flag_x,
                           int *loop_count, int *buff_cycle, int *fix_buff_size ) {

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-nx") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -nx without a value.\n");
            }

            exit(-1);
        }

        *nx = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-ny") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -ny without a value.\n");
            }

            exit(-1);
        }

        *ny = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-nz") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -nz without a value.\n");
            }

            exit(-1);
        }

        *nz = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pex") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pex without a value.\n");
            }

            exit(-1);
        }

        *pex = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pey") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pey without a value.\n");
            }

            exit(-1);
        }

        *pey = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-pez") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -pez without a value.\n");
            }

            exit(-1);
        }

        *pez = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-iterations") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -iterations without a value.\n");
            }

            exit(-1);
        }

        *repeats = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-vars") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -vars without a value.\n");
            }

            exit(-1);
        }

        *vars = atoi(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-sleep") == 0) {
        if (i == argc) {
            if (myrank == 0) {
            fprintf(stderr, "Error: specified -sleep without a value.\n");
            }

            exit(-1);
        }

        *sleep = atol(argv[i + 1]);
        ++i;
        } else if (strcmp(argv[i], "-l") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                    fprintf(stderr, "Error: specified -l without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_l = 1;
            *loop_count = atoi(argv[i + 1]);
            if (*loop_count <= 0) {
                fprintf(stderr, "Error: loop_count must be a positive integer.\n");
                exit(__LINE__);
            }
            i++;
        } else if (strcmp(argv[i], "-b") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                    fprintf(stderr, "Error: specified -b without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_b = 1;
            *buff_cycle = atoi(argv[i + 1]);
            if (*buff_cycle <= 0) {
                fprintf(stderr, "Error: buff_cycle must be a positive integer.\n");
                exit(__LINE__);
            }
            i++;
        } else if (strcmp(argv[i], "-x") == 0) {
            if (i == argc) {
                if (myrank == 0) {
                fprintf(stderr, "Error: specified -x without a value.\n");
                }

                exit(__LINE__);
            }

            *flag_x = 1;
            *fix_buff_size = atoi(argv[i + 1]);
            if (*fix_buff_size < 0) {
                fprintf(stderr, "Error: fixed buff_size must be >= 0.\n");
                exit(__LINE__);
            }

            i++;
        } else {
        if (0 == myrank) {
            fprintf(stderr, "Unknown option: %s\n", argv[i]);
        }

        exit(-1);
        }
    }
}

// HB --> Host_buffer, DB --> Device_buffer, DT --> data type, SZ --> size
#ifdef PINNED
#define INIT_HALO3D_BUFFER(HB, DB, DT, SZ) {                \
    int rank; MPI_Comm_rank(MPI_COMM_WORLD, &rank);         \
    hipHostAlloc(&HB, sizeof(DT) * SZ, hipHostMallocDefault);                    \
    cudaErrorCheck( hipMalloc(&DB, sizeof(DT) * SZ) );     \
    cudaErrorCheck( hipMemset(DB, 0, sizeof(DT) * SZ) );   \
    cudaErrorCheck( hipDeviceSynchronize() );              \
}

#define FREE_HALO3D_BUFFER(HB, DB) {                \
    cudaErrorCheck( hipFree(DB) );                 \
    hipHostFree(HB);                               \
}
#else
#define INIT_HALO3D_BUFFER(HB, DB, DT, SZ) {                \
    int rank; MPI_Comm_rank(MPI_COMM_WORLD, &rank);         \
    HB = (DT*)malloc(sizeof(DT) * SZ);                      \
    cudaErrorCheck( hipMalloc(&DB, sizeof(DT) * SZ) );     \
    cudaErrorCheck( hipMemset(DB, 0, sizeof(DT) * SZ) );   \
    cudaErrorCheck( hipDeviceSynchronize() );              \
}

#define FREE_HALO3D_BUFFER(HB, DB) {                \
    cudaErrorCheck( hipFree(DB) );                 \
    free(HB);                                       \
}
#endif

#define BLK_SIZE 256
#define GRD_SIZE 4
#define TID_DIGITS 10000

__global__
void init_kernel(int n, dtype *input, int rank) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  if (tid < n)
      input[tid] = (dtype)(rank+1);

}

void halo3d_run_axes(int BufferSize, int UpFlag, int DownFlag,
                     dtype *host_UpSendBuffer,   dtype *dev_UpSendBuffer,   dtype *host_UpRecvBuffer,   dtype *dev_UpRecvBuffer,
                     dtype *host_DownSendBuffer, dtype *dev_DownSendBuffer, dtype *host_DownRecvBuffer, dtype *dev_DownRecvBuffer,
                     double *timeTakenCUDA, double *timeTakenMPI, int tag) {
    TIMER_DEF(0);

    int requestcount = 0;
    MPI_Status* status;
    status = (MPI_Status*)malloc(sizeof(MPI_Status) * 4);

    MPI_Request* requests;
    requests = (MPI_Request*)malloc(sizeof(MPI_Request) * 4);
    // =================================================================================================================

    TIMER_START(0);
    if (UpFlag > -1) {
      MPI_Irecv(dev_UpRecvBuffer, BufferSize, MPI_dtype, UpFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(dev_UpSendBuffer, BufferSize, MPI_dtype, UpFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
    }

    if (DownFlag > -1) {
      MPI_Irecv(dev_DownRecvBuffer, BufferSize, MPI_dtype, DownFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(dev_DownSendBuffer, BufferSize, MPI_dtype, DownFlag, tag,
                MPI_COMM_WORLD, &requests[requestcount++]);
    }

    MPI_Waitall(requestcount, requests, status);
    requestcount = 0;
    TIMER_STOP(0);
    *timeTakenMPI += TIMER_ELAPSED(0);

    // =================================================================================================================
}

unsigned int check_recv_buffer (int my_rank, char axe,
                                int UpFlag, dtype *dev_UpBuffer,
                                int DownFlag, dtype *dev_DownBuffer,
                                int BufferSize) {

    cktype UpCheck = 0, DownCheck = 0;
    if (UpFlag>-1) gpu_device_reduce(dev_UpBuffer, BufferSize, &UpCheck);
    if (DownFlag>-1) gpu_device_reduce(dev_DownBuffer, BufferSize, &DownCheck);

    unsigned int result = 0U;
    if ( UpFlag>-1 && UpCheck != BufferSize*(UpFlag+1) ) result |= 1U;
    if ( DownFlag>-1 && DownCheck != BufferSize*(DownFlag+1) ) result |= 2U;
//     printf("[BufferSize=%d, myRank=%d, axe=%c] UpFlag = %d, UpCheck = %d, DownFlag = %d, DownCheck = %d --> %u\n", BufferSize, my_rank, axe, UpFlag, UpCheck, DownFlag, DownCheck, result);
    return(result);
}

// ----------------------------------------------------------------------------


int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */




    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(host_name, &namelen);
    MPI_Barrier(MPI_COMM_WORLD);

    printf("Size = %d, myrank = %d, host_name = %s\n", size, rank, host_name);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Status stat;

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    // print device affiniy
#ifndef SKIPCPUAFFINITY
    if (0==rank) printf("List device affinity:\n");
    check_cpu_and_gpu_affinity(dev);
    if (0==rank) printf("List device affinity done.\n\n");
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);

    int rank2 = size-1;
    MPI_Barrier(MPI_COMM_WORLD);

    /* -------------------------------------------------------------------------------------------
        Reading command line inputs
    --------------------------------------------------------------------------------------------*/

    int opt;
    int max_j;
    int flag_b = 0;
    int flag_l = 0;
    int flag_x = 0;
    int loop_count = LOOP_COUNT;
    int buff_cycle = BUFF_CYCLE;
    int fix_buff_size = 0;

    // Define halo3D parameters
    int pex = size, pey = 1, pez = 1;
    int nx = 10, ny = 10, nz = 10;
    long sleep = 1000;
    int repeats = 100;
    int vars = 1;

    // Set default 3D grid
    {
        int M=0, K=0, H=0;
        int n=0, k=0, h=0;
        while ( ((size)%(1<<(n+1))) == 0 ) n++;

        k = n/3;
        h = (n - k)/2;

        printf("n = %d --> k = %d, h = %d\n", n, k, h);

        K = 1 << k;
        H = 1 << h;
        M = size / (1 << (k+h));

        printf("size = %d --> %d x %d x %d\n", size, M, H, K);

        pex = M;
        pey = H;
        pez = K;
    }

    // Read input parameters
    read_line_parameters(argc, argv, rank, &nx, &ny, &nz, &pex, &pey, &pez, &repeats, &vars, &sleep,
                                           &flag_b, &flag_l, &flag_x, &loop_count, &buff_cycle, &fix_buff_size);
    MPI_Barrier(MPI_COMM_WORLD);
    if(flag_x && fix_buff_size >= buff_cycle){buff_cycle = fix_buff_size + 1;}    
    // Print message based on the flags
    if (flag_b && rank == 0) printf("Flag b was set with argument: %d\n", buff_cycle);
    if (flag_l && rank == 0) printf("Flag l was set with argument: %d\n", loop_count);
    if (flag_x && rank == 0) printf("Flag x was set with argument: %d\n", fix_buff_size);

    max_j = (flag_x == 0) ? buff_cycle : (fix_buff_size + 1) ;
    if (rank == 0) printf("buff_cycle: %d loop_count: %d max_j: %d\n", buff_cycle, loop_count, max_j);
    if (flag_x > 0 && rank == 0) printf("fix_buff_size is set as %d\n", fix_buff_size);

    /* -------------------------------------------------------------------------------------------
        Halo 3D Initialization
    --------------------------------------------------------------------------------------------*/

    // Check for correct phisical initizlization
    if ((pex * pey * pez) != size) {
        if (0 == rank) {
        fprintf(stderr, "Error: rank grid does not equal number of ranks.\n");
        fprintf(stderr, "%7d x %7d x %7d != %7d\n", pex, pey, pez, size);
        }

        exit(-1);
    }
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    // Print data
    if (rank == 0) {
        printf("# MPI Nearest Neighbor Communication\n");
        printf("# Info:\n");
        printf("# Processor Grid:         %7d x %7d x %7d\n", pex, pey, pez);
        printf("# Data Grid (per rank):   %7d x %7d x %7d\n", nx, ny, nz);
        printf("# Iterations:             %7d\n", repeats);
        printf("# Variables:              %7d\n", vars);
        printf("# Sleep:                  %7ld\n", sleep);
    }
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    // Compute position and flags
    int posX, posY, posZ;
    get_position(rank, pex, pey, pez, &posX, &posY, &posZ);

    int xUp = convert_position_to_rank(pex, pey, pez, posX + 1, posY, posZ);
    int yUp = convert_position_to_rank(pex, pey, pez, posX, posY + 1, posZ);
    int zUp = convert_position_to_rank(pex, pey, pez, posX, posY, posZ + 1);
    int xDown = convert_position_to_rank(pex, pey, pez, posX - 1, posY, posZ);
    int yDown = convert_position_to_rank(pex, pey, pez, posX, posY - 1, posZ);
    int zDown = convert_position_to_rank(pex, pey, pez, posX, posY, posZ - 1);

    // Declare buffers and sizes variables
    size_t xSize, ySize, zSize;

    dtype     *xUpSendBuffer,     *xUpRecvBuffer,     *xDownSendBuffer,     *xDownRecvBuffer;
    dtype *dev_xUpSendBuffer, *dev_xUpRecvBuffer, *dev_xDownSendBuffer, *dev_xDownRecvBuffer;

    dtype     *yUpSendBuffer,     *yUpRecvBuffer,     *yDownSendBuffer,     *yDownRecvBuffer;
    dtype *dev_yUpSendBuffer, *dev_yUpRecvBuffer, *dev_yDownSendBuffer, *dev_yDownRecvBuffer;

    dtype     *zUpSendBuffer,     *zUpRecvBuffer,     *zDownSendBuffer,     *zDownRecvBuffer;
    dtype *dev_zUpSendBuffer, *dev_zUpRecvBuffer, *dev_zDownSendBuffer, *dev_zDownRecvBuffer;


     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    SZTYPE N;
    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    double start_time, stop_time;
    double cuda_timer[3], mpi_timer[3];
    double *elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    double *inner_elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    unsigned int *halo_checks = (unsigned int*)malloc(sizeof(unsigned int)*buff_cycle);
    for(int j=fix_buff_size; j<max_j; j++){

        // Define cycle sizes
        (j!=0) ? (N <<= 1) : (N = 1);
        xSize = ny * nz * N;
        ySize = nx * nz * N;
        zSize = nx * ny * N;
        halo_checks[j] = 0U;

//         STR_COLL_DEF
//         STR_COLL_INIT

        // Alloc x axe
        INIT_HALO3D_BUFFER(xUpSendBuffer, dev_xUpSendBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xUpRecvBuffer, dev_xUpRecvBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xDownSendBuffer, dev_xDownSendBuffer, dtype, xSize)
        INIT_HALO3D_BUFFER(xDownRecvBuffer, dev_xDownRecvBuffer, dtype, xSize)

        // Alloc y axe
        INIT_HALO3D_BUFFER(yUpSendBuffer, dev_yUpSendBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yUpRecvBuffer, dev_yUpRecvBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yDownSendBuffer, dev_yDownSendBuffer, dtype, ySize)
        INIT_HALO3D_BUFFER(yDownRecvBuffer, dev_yDownRecvBuffer, dtype, ySize)

        // Alloc z axe
        INIT_HALO3D_BUFFER(zUpSendBuffer, dev_zUpSendBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zUpRecvBuffer, dev_zUpRecvBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zDownSendBuffer, dev_zDownSendBuffer, dtype, zSize)
        INIT_HALO3D_BUFFER(zDownRecvBuffer, dev_zDownRecvBuffer, dtype, zSize)

//         MPI_ALL_PRINT(fprintf(fp, "%s", STR_COLL_GIVE);)
//         STR_COLL_FREE

        cudaErrorCheck( hipDeviceSynchronize() );
        MPI_Barrier(MPI_COMM_WORLD);
        fflush(stdout);

        // Init send buffers (Recv buffers stay initialized as 0)
        {
            size_t maxSize = (xSize > ySize) ? xSize : ySize;
            if (zSize > maxSize) maxSize = zSize;
            size_t run_time_grid_size = (maxSize % BLK_SIZE == 0) ? (maxSize/BLK_SIZE) : ((maxSize/BLK_SIZE) +1);
            dim3 block_size(BLK_SIZE, 1, 1);
            dim3 grid_size(run_time_grid_size, 1, 1);
            init_kernel<<<grid_size, block_size>>>(xSize, dev_xUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(ySize, dev_yUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(zSize, dev_zUpSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(xSize, dev_xDownSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(ySize, dev_yDownSendBuffer, rank);
            init_kernel<<<grid_size, block_size>>>(zSize, dev_zDownSendBuffer, rank);
            cudaErrorCheck( hipDeviceSynchronize() );
            MPI_Barrier(MPI_COMM_WORLD);
        }

        if (rank == 0) {printf("%d#", j); fflush(stdout);}
        cudaErrorCheck( hipDeviceSynchronize() );
        MPI_Barrier(MPI_COMM_WORLD);
        fflush(stdout);

        /*

        Implemetantion goes here

        */

        for(int i=1-(WARM_UP); i<=loop_count; i++) {
            for (int k=0; k<3; k++) {cuda_timer[k] = 0.0; mpi_timer[k] = 0.0;}
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

            halo3d_run_axes(xSize, xUp, xDown,
                     NULL, dev_xUpSendBuffer,   NULL, dev_xUpRecvBuffer,
                     NULL, dev_xDownSendBuffer, NULL, dev_xDownRecvBuffer,
                     &(cuda_timer[0]), &(mpi_timer[0]), 1000);

            halo3d_run_axes(ySize, yUp, yDown,
                     NULL, dev_yUpSendBuffer,   NULL, dev_yUpRecvBuffer,
                     NULL, dev_yDownSendBuffer, NULL, dev_yDownRecvBuffer,
                     &(cuda_timer[1]), &(mpi_timer[1]), 2000);

            halo3d_run_axes(zSize, zUp, zDown,
                     NULL, dev_zUpSendBuffer,   NULL, dev_zUpRecvBuffer,
                     NULL, dev_zDownSendBuffer, NULL, dev_zDownRecvBuffer,
                     &(cuda_timer[2]), &(mpi_timer[2]), 3000);

            stop_time = MPI_Wtime();
            if (i>0) inner_elapsed_time[(j-fix_buff_size)*loop_count+i-1] = stop_time - start_time;

            if (rank == 0) {printf("%%"); fflush(stdout);}

            unsigned int xCheck = check_recv_buffer(rank, 'x', xUp, dev_xUpRecvBuffer, xDown, dev_xDownRecvBuffer, xSize);
            unsigned int yCheck = check_recv_buffer(rank, 'y', yUp, dev_yUpRecvBuffer, yDown, dev_yDownRecvBuffer, ySize);
            unsigned int zCheck = check_recv_buffer(rank, 'z', zUp, dev_zUpRecvBuffer, zDown, dev_zDownRecvBuffer, zSize);
            xCheck = xCheck << 4;
            yCheck = yCheck << 2;
            halo_checks[j] |= xCheck;
            halo_checks[j] |= yCheck;
            halo_checks[j] |= zCheck;
        }
        if (rank == 0) {printf("#\n"); fflush(stdout);}


        // Free x axe
        FREE_HALO3D_BUFFER(xUpSendBuffer, dev_xUpSendBuffer)
        FREE_HALO3D_BUFFER(xUpRecvBuffer, dev_xUpRecvBuffer)
        FREE_HALO3D_BUFFER(xDownSendBuffer, dev_xDownSendBuffer)
        FREE_HALO3D_BUFFER(xDownRecvBuffer, dev_xDownRecvBuffer)

        // Free y axe
        FREE_HALO3D_BUFFER(yUpSendBuffer, dev_yUpSendBuffer)
        FREE_HALO3D_BUFFER(yUpRecvBuffer, dev_yUpRecvBuffer)
        FREE_HALO3D_BUFFER(yDownSendBuffer, dev_yDownSendBuffer)
        FREE_HALO3D_BUFFER(yDownRecvBuffer, dev_yDownRecvBuffer)

        // Free z axe
        FREE_HALO3D_BUFFER(zUpSendBuffer, dev_zUpSendBuffer)
        FREE_HALO3D_BUFFER(zUpRecvBuffer, dev_zUpRecvBuffer)
        FREE_HALO3D_BUFFER(zDownSendBuffer, dev_zDownSendBuffer)
        FREE_HALO3D_BUFFER(zDownRecvBuffer, dev_zDownRecvBuffer)
    }

    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    MPI_Allreduce(inner_elapsed_time, elapsed_time, buff_cycle*loop_count, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    for(int j=fix_buff_size; j<max_j; j++) {
        (j!=0) ? (N <<= 1) : (N = 1);

        SZTYPE num_B, int_num_GB;
        double num_GB;

        if (j < 31) {
            SZTYPE B_in_GB = 1 << 30;
            num_B = sizeof(dtype)*N*((size-1)/(float)size)*2;
            num_GB = (double)num_B / (double)B_in_GB;
        } else {
            SZTYPE M = 1 << (j - 30);
            num_B = N*((size-1)/(float)size)*2*sizeof(dtype);
            num_GB = sizeof(dtype)*M*((size-1)/(float)size)*2;
        }

        if (xUp > -1) num_B += ny * nz * N;
        if (yUp > -1) num_B += nx * nz * N;
        if (zUp > -1) num_B += nx * ny * N;
        if (xDown > -1) num_B += ny * nz * N;
        if (yDown > -1) num_B += nx * nz * N;
        if (zDown > -1) num_B += nx * ny * N;

        double avg_time_per_transfer = 0.0;
        for (int i=0; i<loop_count; i++) {
            avg_time_per_transfer += elapsed_time[(j-fix_buff_size)*loop_count+i];
            if(rank == 0) printf("\tTransfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Iteration %d\n", num_B, elapsed_time[(j-fix_buff_size)*loop_count+i], num_GB/elapsed_time[(j-fix_buff_size)*loop_count+i], i);
        }
        avg_time_per_transfer /= ((double)loop_count);

        if(rank == 0) printf("[Average] Transfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, halo_checks[j] );
        fflush(stdout);
    }

    free(halo_checks);
    free(elapsed_time);
    free(inner_elapsed_time);
    MPI_Finalize();
    return(0);
}
