#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>
#include <unistd.h>
#include <inttypes.h>

#define MPI

#include "../include/error.h"
#include "../include/type.h"
#include "../include/gpu_ops.h"
#include "../include/device_assignment.h"
#include "../include/cmd_util.h"
#include "../include/prints.h"

#ifdef MPIX_CUDA_AWARE_SUPPORT
/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"
#endif

#define BUFF_CYCLE 28
#define LOOP_COUNT 50

#define WARM_UP 5

int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */




    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(host_name, &namelen);
    MPI_Barrier(MPI_COMM_WORLD);

    printf("Size = %d, myrank = %d, host_name = %s\n", size, rank, host_name);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    // print device affiniy
#ifndef SKIPCPUAFFINITY
    if (0==rank) printf("List device affinity:\n");
    check_cpu_and_gpu_affinity(dev);
    if (0==rank) printf("List device affinity done.\n\n");
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);


    /* -------------------------------------------------------------------------------------------
        Reading command line inputs
    --------------------------------------------------------------------------------------------*/

    int opt;
    int max_j;
    int flag_b = 0;
    int flag_l = 0;
    int flag_x = 0;
    int loop_count = LOOP_COUNT;
    int buff_cycle = BUFF_CYCLE;
    int fix_buff_size = 0;

    // Parse command-line options
    read_line_parameters(argc, argv, rank,
                         &flag_b, &flag_l, &flag_x,
                         &loop_count, &buff_cycle, &fix_buff_size);
    if(flag_x && fix_buff_size >= buff_cycle){buff_cycle = fix_buff_size + 1;}    
    // Print message based on the flags
    if (flag_b && rank == 0) printf("Flag b was set with argument: %d\n", buff_cycle);
    if (flag_l && rank == 0) printf("Flag l was set with argument: %d\n", loop_count);
    if (flag_x && rank == 0) printf("Flag x was set with argument: %d\n", fix_buff_size);

    max_j = (flag_x == 0) ? buff_cycle : (fix_buff_size + 1) ;
    if (rank == 0) printf("buff_cycle: %d loop_count: %d max_j: %d\n", buff_cycle, loop_count, max_j);
    if (flag_x > 0 && rank == 0) printf("fix_buff_size is set as %d\n", fix_buff_size);

    // Check if I am one of the destination ranks
    // Read the GPUBENCH_OTOM_DEST environment variable (comma-separated string of ranks)    
    char *dest_ranks_str = getenv("GPUBENCH_OTOM_DEST");
    size_t num_destinations;
    int* dest_ranks = (int*) malloc(size*sizeof(int));
    if (dest_ranks_str != NULL) {
        memset(dest_ranks, 0, size*sizeof(int));
        num_destinations = 0;
        char *token = strtok(dest_ranks_str, ",");
        while (token != NULL) {
            dest_ranks[atoi(token)] = 1;
            ++num_destinations;
	        token = strtok(NULL, ",");
        }
    }else{
        memset(dest_ranks, 1, size*sizeof(int));
        num_destinations = size - 1;
    }

     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    SZTYPE N;
    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    double start_time, stop_time;
    int *error = (int*)malloc(sizeof(int)*buff_cycle);
    int *my_error = (int*)malloc(sizeof(int)*buff_cycle);
    cktype *cpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    cktype *gpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    double *elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    double *inner_elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    for(int j=fix_buff_size; j<max_j; j++){

        (j!=0) ? (N <<= 1) : (N = 1);
        if (rank == 0) {printf("%i#", j); fflush(stdout);}

        // Allocate memory for A on CPU
        dtype *A, *B;
#ifdef PINNED
        hipHostAlloc(&A, size*N*sizeof(dtype), hipHostMallocDefault);
        hipHostAlloc(&B, size*N*sizeof(dtype), hipHostMallocDefault);
#else
        A = (dtype*)malloc(size*N*sizeof(dtype));
        B = (dtype*)malloc(size*N*sizeof(dtype));
#endif
        cktype *my_cpu_check = (cktype*)malloc(sizeof(cktype)*size);
        cktype *recv_cpu_check = (cktype*)malloc(sizeof(cktype)*size), gpu_check = 0;
        for (int i=0; i<size; i++)
            my_cpu_check[i] = 0U;

        // Initialize all elements of A to 0.0
        for(SZTYPE i=0; i<N*size; i++) {
            A[i] = 1U * (rank+1);
            B[i] = 0U;
        }

        dtype *d_B;
        cudaErrorCheck( hipMalloc(&d_B, size*N*sizeof(dtype)) );
        cudaErrorCheck( hipMemcpy(d_B, B, size*N*sizeof(dtype), hipMemcpyHostToDevice) );

        dtype *d_A;
        cudaErrorCheck( hipMalloc(&d_A, size*N*sizeof(dtype)) );
        cudaErrorCheck( hipMemcpy(d_A, A, size*N*sizeof(dtype), hipMemcpyHostToDevice) );

        for (int i=0; i<size; i++)
            gpu_device_reduce(d_A + (i*N)*sizeof(dtype), N, &my_cpu_check[i]);

        /*

        Implemetantion goes here

        */
        MPI_Request* requests = (MPI_Request*) malloc(sizeof(MPI_Request)*num_destinations);
        for(int i=1-(WARM_UP); i<=loop_count; i++){
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

            // Assume the root of the otom is rank 0
            if(rank == 0){
                int next_req = 0;
                for (int r=1; r<size; r++){
                    if(dest_ranks[r]){
                        MPI_Isend(d_A, N, MPI_dtype, r, 0, MPI_COMM_WORLD, &requests[next_req]);
                        ++next_req;
                    }
                }
                MPI_Waitall(num_destinations, requests, MPI_STATUSES_IGNORE);
            }else if(dest_ranks[rank]){
                MPI_Recv(d_A, N, MPI_dtype, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            }

            stop_time = MPI_Wtime();
            if (i>0) inner_elapsed_time[(j-fix_buff_size)*loop_count+i-1] = stop_time - start_time;

            if (rank == 0) {printf("%%"); fflush(stdout);}
        }
        free(requests);
        if (rank == 0) {printf("#\n"); fflush(stdout);}




        gpu_device_reduce(d_B, size*N, &gpu_check);
        MPI_Alltoall(my_cpu_check, 1, MPI_cktype, recv_cpu_check, 1, MPI_cktype, MPI_COMM_WORLD);

        cpu_checks[j] = 0;
        gpu_checks[j] = gpu_check;
        for (int i=0; i<size; i++)
            cpu_checks[j] += recv_cpu_check[i];
        my_error[j] = abs(gpu_checks[j] - cpu_checks[j]);

        cudaErrorCheck( hipFree(d_A) );
        cudaErrorCheck( hipFree(d_B) );
        free(recv_cpu_check);
        free(my_cpu_check);
#ifdef PINNED
        hipHostFree(A);
        hipHostFree(B);
#else
        free(A);
        free(B);
#endif
    }

    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    MPI_Allreduce(my_error, error, buff_cycle, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(inner_elapsed_time, elapsed_time, buff_cycle*loop_count, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    for(int j=fix_buff_size; j<max_j; j++) {
        (j!=0) ? (N <<= 1) : (N = 1);

        SZTYPE num_B, int_num_GB;
        double num_GB;

        num_B = sizeof(dtype)*N*num_destinations;
        // TODO: maybe we can avoid if and just divide always by B_in_GB
        if (j < 31) {
            SZTYPE B_in_GB = 1 << 30;
            num_GB = (double)num_B / (double)B_in_GB;
        } else {
            SZTYPE M = 1 << (j - 30);            
            num_GB = sizeof(dtype)*M*num_destinations;
        }

        double avg_time_per_transfer = 0.0;
        for (int i=0; i<loop_count; i++) {
            avg_time_per_transfer += elapsed_time[(j-fix_buff_size)*loop_count+i];
            if(rank == 0) printf("\tTransfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Iteration %d\n", num_B, elapsed_time[(j-fix_buff_size)*loop_count+i], num_GB/elapsed_time[(j-fix_buff_size)*loop_count+i], i);
        }
        avg_time_per_transfer /= ((double)loop_count);

        if(rank == 0) printf("[Average] Transfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, error[j] );
        fflush(stdout);
    }

    char *s = (char*)malloc(sizeof(char)*(20*buff_cycle + 100));
    sprintf(s, "[%d] recv_cpu_check = %u", rank, cpu_checks[0]);
    for (int i=fix_buff_size; i<max_j; i++) {
        sprintf(s+strlen(s), " %10d", cpu_checks[i]);
    }
    sprintf(s+strlen(s), " (for Error)\n");
    printf("%s", s);
    fflush(stdout);

    sprintf(s, "[%d] gpu_checks = %u", rank, gpu_checks[0]);
    for (int i=fix_buff_size; i<max_j; i++) {
        sprintf(s+strlen(s), " %10d", gpu_checks[i]);
    }
    sprintf(s+strlen(s), " (for Error)\n");
    printf("%s", s);
    fflush(stdout);

    free(error);
    free(my_error);
    free(cpu_checks);
    free(gpu_checks);
    free(elapsed_time);
    free(inner_elapsed_time);
    MPI_Finalize();
    return(0);
}
