#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>
#include <unistd.h>
#include <inttypes.h>

#define MPI

#include "../include/error.h"
#include "../include/type.h"
#include "../include/gpu_ops.h"
#include "../include/device_assignment.h"
#include "../include/cmd_util.h"
#include "../include/prints.h"

#ifdef MPIX_CUDA_AWARE_SUPPORT
/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"
#endif

#define BUFF_CYCLE 31
#define LOOP_COUNT 50

#define WARM_UP 5

// ---------------------------------------
void PICO_enable_peer_access(int myrank, int deviceCount, int mydev) {
    // Pick all the devices that can access each other's memory for this test
    // Keep in mind that CUDA has minimal support for fork() without a
    // corresponding exec() in the child process, but in this case our
    // spawnProcess will always exec, so no need to worry.
    hipDeviceProp_t prop;
    int allPeers = 1, myIPC = 1, allIPC;
    cudaErrorCheck(hipGetDeviceProperties(&prop, mydev));

    int* canAccesPeer = (int*) malloc(sizeof(int)*deviceCount*deviceCount);
    for (int i = 0; i < deviceCount*deviceCount; i++) canAccesPeer[i] = 0;

    // CUDA IPC is only supported on devices with unified addressing
    if (!prop.unifiedAddressing) {
      myIPC = 0;
    } else {
    }
    // This sample requires two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      myIPC = 0;
    }

    MPI_Allreduce(&myIPC, &allIPC, 1, MPI_INT, MPI_MIN, MPI_COMM_WORLD);
    if (!allIPC) {
      exit(__LINE__);
    }

    if (myrank == 0) {
      for (int i = 0; i < deviceCount; i++) {
        for (int j = 0; j < deviceCount; j++) {
          if (j != i) {
            int canAccessPeerIJ, canAccessPeerJI;
            cudaErrorCheck( hipDeviceCanAccessPeer(&canAccessPeerJI, j, i) );
            cudaErrorCheck( hipDeviceCanAccessPeer(&canAccessPeerIJ, i, j) );

            canAccesPeer[i * deviceCount + j] = (canAccessPeerIJ) ? 1 : 0;
            canAccesPeer[j * deviceCount + i] = (canAccessPeerJI) ? 1 : 0;
            if (!canAccessPeerIJ || !canAccessPeerJI) allPeers = 0;
          } else {
            canAccesPeer[i * deviceCount + j] = -1;
          }
        }
      }
    }

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&allPeers, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(canAccesPeer, deviceCount*deviceCount, MPI_INT, 0, MPI_COMM_WORLD);

    if (allPeers) {
      // Enable peers here.  This isn't necessary for IPC, but it will
      // setup the peers for the device.  For systems that only allow 8
      // peers per GPU at a time, this acts to remove devices from CanAccessPeer
      for (int j = 0; j < deviceCount; j++) {
        if (j != mydev) {
          cudaErrorCheck(hipDeviceEnablePeerAccess(j, 0));
        }
      }
    }

    MPI_Barrier(MPI_COMM_WORLD);
}

void PICO_disable_peer_access(int deviceCount, int mydev){
    MPI_Barrier(MPI_COMM_WORLD);
    for (int j = 0; j < deviceCount; j++) {
      if (j != mydev) {
        cudaErrorCheck(hipDeviceDisablePeerAccess(j));
      }
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(host_name, &namelen);
    MPI_Barrier(MPI_COMM_WORLD);

    printf("Size = %d, myrank = %d, host_name = %s\n", size, rank, host_name);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Status stat;

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    // print device affiniy
#ifndef SKIPCPUAFFINITY
    if (0==rank) printf("List device affinity:\n");
    check_cpu_and_gpu_affinity(dev);
    if (0==rank) printf("List device affinity done.\n\n");
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);

//     if (nnodes != 1) {
//         if (0 == rank) printf("The NVLINK version is only implemented for intraNode communication\n");
//         exit(__LINE__);
//     }

    int rank2 = size-1;

    // Get the group or processes of the default communicator
    MPI_Group world_group;
    MPI_Comm_group(MPI_COMM_WORLD, &world_group);

    // Keep only the processes 0 and 1 in the new group.
    int ranks[2];
    ranks[0] = 0;
    ranks[1] = rank2;
    MPI_Group pp_group;
    MPI_Group_incl(world_group, 2, ranks, &pp_group);

    // Create the new communicator from that group of processes.
    MPI_Comm ppComm;
    MPI_Comm_create(MPI_COMM_WORLD, pp_group, &ppComm);

    // Do a broadcast only between the processes of the new communicator.

    if(ppComm == MPI_COMM_NULL) {
        // I am not part of the ppComm.
        printf("Process %d did not take part to the ppComm.\n", rank);
    } else {
        // I am part of the new ppComm.
        printf("Process %d took part to the ppComm.\n", rank);
    }

    // Keep only the first sender processe (i.e. 0) in the new group.
    int ranks_0[1];
    ranks_0[0] = 0;
    MPI_Group firstsender_group;
    if (rank == 0)
        MPI_Group_incl(world_group, 1, ranks_0, &firstsender_group);
    else
        MPI_Group_incl(world_group, 0, ranks_0, &firstsender_group);

    // Create the new communicator from that group of processes.
    MPI_Comm firstsenderComm;
    MPI_Comm_create(MPI_COMM_WORLD, firstsender_group, &firstsenderComm);

    // Do a broadcast only between the processes of the new communicator.

    if(firstsenderComm == MPI_COMM_NULL) {
        printf("Process %d did not take part to the firstsenderComm.\n", rank);
    } else {
        printf("Process %d took part to the firstsenderComm.\n", rank);
    }
    MPI_Barrier(MPI_COMM_WORLD);

    /* -------------------------------------------------------------------------------------------
        Reading command line inputs
    --------------------------------------------------------------------------------------------*/

    int opt;
    int max_j;
    int flag_b = 0;
    int flag_l = 0;
    int flag_x = 0;
    int loop_count = LOOP_COUNT;
    int buff_cycle = BUFF_CYCLE;
    int fix_buff_size = 0;

    // Parse command-line options
    read_line_parameters(argc, argv, rank,
                         &flag_b, &flag_l, &flag_x,
                         &loop_count, &buff_cycle, &fix_buff_size);
    if(flag_x && fix_buff_size >= buff_cycle){buff_cycle = fix_buff_size + 1;}    
    // Print message based on the flags
    if (flag_b && rank == 0) printf("Flag b was set with argument: %d\n", buff_cycle);
    if (flag_l && rank == 0) printf("Flag l was set with argument: %d\n", loop_count);
    if (flag_x && rank == 0) printf("Flag x was set with argument: %d\n", fix_buff_size);

    max_j = (flag_x == 0) ? buff_cycle : (fix_buff_size + 1) ;
    if (rank == 0) printf("buff_cycle: %d loop_count: %d max_j: %d\n", buff_cycle, loop_count, max_j);
    if (flag_x > 0 && rank == 0) printf("fix_buff_size is set as %d\n", fix_buff_size);

     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    PICO_enable_peer_access(rank, num_devices, dev);

    SZTYPE N;
    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    double start_time, stop_time;
    int *error = (int*)malloc(sizeof(int)*buff_cycle);
    int *my_error = (int*)malloc(sizeof(int)*buff_cycle);
    cktype *cpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    cktype *gpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    double *elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    double *inner_elapsed_time = (double*)malloc(sizeof(double)*buff_cycle*loop_count);
    if (rank == 0 || rank == rank2) {

        MPI_Status IPCstat;
        dtype *peerBuffer;
        hipEvent_t event;
        hipIpcMemHandle_t sendHandle, recvHandle;

        for(int j=fix_buff_size; j<max_j; j++){

            (j!=0) ? (N <<= 1) : (N = 1);
            if (rank == 0) {printf("%i#", j); fflush(stdout);}

            // Allocate memory for A on CPU
            dtype *A, *B;
            cktype my_cpu_check = 0, recv_cpu_check, gpu_check = 0;
#ifdef PINNED
            hipHostAlloc(&A, N*sizeof(dtype), hipHostMallocDefault);
            hipHostAlloc(&B, N*sizeof(dtype), hipHostMallocDefault);
#else
            A = (dtype*)malloc(N*sizeof(dtype));
            B = (dtype*)malloc(N*sizeof(dtype));
#endif

            // Initialize all elements of A to 0.0
            for(SZTYPE i=0; i<N; i++){
                A[i] = 1U * (rank+1);
                B[i] = 0U;
            }

            dtype *d_B;
            cudaErrorCheck( hipMalloc(&d_B, N*sizeof(dtype)) );
            cudaErrorCheck( hipMemcpy(d_B, B, N*sizeof(dtype), hipMemcpyHostToDevice) );

            dtype *d_A;
            cudaErrorCheck( hipMalloc(&d_A, N*sizeof(dtype)) );
            cudaErrorCheck( hipMemcpy(d_A, A, N*sizeof(dtype), hipMemcpyHostToDevice) );
            gpu_device_reduce(d_A, N, &my_cpu_check);

            int tag1 = 10;
            int tag2 = 20;

            /*

            Implemetantion goes here

            */
            // Generate IPC MemHandle
            cudaErrorCheck( hipIpcGetMemHandle((hipIpcMemHandle_t*)&sendHandle, d_A) );

            // Share IPC MemHandle
            if (rank == 0) {
                MPI_Send(&sendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, rank2, 0, MPI_COMM_WORLD);
                MPI_Recv(&recvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, rank2, 1, MPI_COMM_WORLD, &IPCstat);
            }
            if (rank == rank2) {
                MPI_Recv(&recvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, 0, MPI_COMM_WORLD, &IPCstat);
                MPI_Send(&sendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, 1, MPI_COMM_WORLD);
            }

            // Open MemHandle
            cudaErrorCheck( hipIpcOpenMemHandle((void**)&peerBuffer, *(hipIpcMemHandle_t*)&recvHandle, hipIpcMemLazyEnablePeerAccess) );


            for(int i=1-(WARM_UP); i<=loop_count; i++){
                MPI_Barrier(ppComm);
                start_time = MPI_Wtime();

                // Memcopy DeviceToDevice
                if (rank == 0) {
                    cudaErrorCheck( hipMemcpy(d_B, peerBuffer, sizeof(dtype)*N, hipMemcpyDeviceToDevice) );
                    cudaErrorCheck( hipDeviceSynchronize() );
                }
                MPI_Barrier(ppComm);
                if (rank == rank2) {
                    cudaErrorCheck( hipMemcpy(d_B, peerBuffer, sizeof(dtype)*N, hipMemcpyDeviceToDevice) );
                    cudaErrorCheck( hipDeviceSynchronize() );
                }
                MPI_Barrier(ppComm);

                stop_time = MPI_Wtime();
                if (i>0) inner_elapsed_time[(j-fix_buff_size)*loop_count+i-1] = stop_time - start_time;

                if (rank == 0) {printf("%%"); fflush(stdout);}
            }
            if (rank == 0) {printf("#\n"); fflush(stdout);}

            // Close MemHandle
            cudaErrorCheck( hipIpcCloseMemHandle(peerBuffer) );



            gpu_device_reduce(d_B, N, &gpu_check);
            if(rank == 0){
                MPI_Send(&my_cpu_check,   1, MPI_cktype, rank2, tag1, MPI_COMM_WORLD);
                MPI_Recv(&recv_cpu_check, 1, MPI_cktype, rank2, tag2, MPI_COMM_WORLD, &stat);
            } else if(rank == rank2){
                MPI_Recv(&recv_cpu_check, 1, MPI_cktype, 0, tag1, MPI_COMM_WORLD, &stat);
                MPI_Send(&my_cpu_check,   1, MPI_cktype, 0, tag2, MPI_COMM_WORLD);
            }

            gpu_checks[j] = gpu_check;
            cpu_checks[j] = recv_cpu_check;
            my_error[j] = abs(gpu_checks[j] - cpu_checks[j]);

            cudaErrorCheck( hipFree(d_A) );
            cudaErrorCheck( hipFree(d_B) );
#ifdef PINNED
            hipHostFree(A);
            hipHostFree(B);
#else
            free(A);
            free(B);
#endif
        }

        if (fix_buff_size<=30) {
            N = 1 << (fix_buff_size - 1);
        } else {
            N = 1 << 30;
            N <<= (fix_buff_size - 31);
        }

        MPI_Allreduce(my_error, error, buff_cycle, MPI_INT, MPI_MAX, ppComm);
        //MPI_Allreduce(inner_elapsed_time, elapsed_time, buff_cycle*loop_count, MPI_DOUBLE, MPI_MAX, firstsenderComm);
        memcpy(elapsed_time, inner_elapsed_time, buff_cycle*loop_count*sizeof(double)); // No need to do allreduce, there is only one rank in firstsenderComm
        for(int j=fix_buff_size; j<max_j; j++) {
            (j!=0) ? (N <<= 1) : (N = 1);

            SZTYPE num_B, int_num_GB;
            double num_GB;

            num_B = sizeof(dtype)*N;
            // TODO: maybe we can avoid if and just divide always by B_in_GB
            if (j < 31) {
                SZTYPE B_in_GB = 1 << 30;
                num_GB = (double)num_B / (double)B_in_GB;
            } else {
                SZTYPE M = 1 << (j - 30);            
                num_GB = sizeof(dtype)*M;
            }

            double avg_time_per_transfer = 0.0;
            for (int i=0; i<loop_count; i++) {
                elapsed_time[(j-fix_buff_size)*loop_count+i] /= 2.0;
                avg_time_per_transfer += elapsed_time[(j-fix_buff_size)*loop_count+i];
                if(rank == 0) printf("\tTransfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Iteration %d\n", num_B, elapsed_time[(j-fix_buff_size)*loop_count+i], num_GB/elapsed_time[(j-fix_buff_size)*loop_count+i], i);
            }
            avg_time_per_transfer /= (double)loop_count;

            if(rank == 0) printf("[Average] Transfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, error[j] );
            fflush(stdout);
        }

        char *s = (char*)malloc(sizeof(char)*(20*buff_cycle + 100));
        sprintf(s, "[%d] recv_cpu_check = %u", rank, cpu_checks[0]);
        for (int i=fix_buff_size; i<max_j; i++) {
            sprintf(s+strlen(s), " %10d", cpu_checks[i]);
        }
        sprintf(s+strlen(s), " (for Error)\n");
        printf("%s", s);
        fflush(stdout);

        sprintf(s, "[%d] gpu_checks = %u", rank, gpu_checks[0]);
        for (int i=fix_buff_size; i<max_j; i++) {
            sprintf(s+strlen(s), " %10d", gpu_checks[i]);
        }
        sprintf(s+strlen(s), " (for Error)\n");
        printf("%s", s);
        fflush(stdout);
    }

    PICO_disable_peer_access(num_devices, dev);

    free(error);
    free(my_error);
    free(cpu_checks);
    free(gpu_checks);
    free(elapsed_time);
    free(inner_elapsed_time);
    MPI_Finalize();
    return(0);
}
