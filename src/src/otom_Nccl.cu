#include "hip/hip_runtime.h"
// One-to-many

#include <stdio.h>
#include "mpi.h"

#include <nccl.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>
#include <unistd.h>
#include <inttypes.h>
#include <chrono>

#define MPI

#include "../include/error.h"
#include "../include/type.h"
#include "../include/gpu_ops.h"
#include "../include/device_assignment.h"
#include "../include/cmd_util.h"
#include "../include/prints.h"

#ifdef MPIX_CUDA_AWARE_SUPPORT
/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"
#endif

#define BUFF_CYCLE 28
#define LOOP_COUNT 50

#define WARM_UP 5

int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */




    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int namelen;
    char host_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(host_name, &namelen);
    MPI_Barrier(MPI_COMM_WORLD);

    printf("Size = %d, myrank = %d, host_name = %s\n", size, rank, host_name);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);


//     if(size != 2){
//         if(rank == 0){
//             printf("This program requires exactly 2 MPI ranks, but you are attempting to use %d! Exiting...\n", size);
//         }
//         MPI_Finalize();
//         exit(0);
//     }

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    // print device affiniy
#ifndef SKIPCPUAFFINITY
    if (0==rank) printf("List device affinity:\n");
    check_cpu_and_gpu_affinity(dev);
    if (0==rank) printf("List device affinity done.\n\n");
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);


    /* -------------------------------------------------------------------------------------------
        Reading command line inputs
    --------------------------------------------------------------------------------------------*/

    int opt;
    int max_j;
    int flag_b = 0;
    int flag_l = 0;
    int flag_x = 0;
    int loop_count = LOOP_COUNT;
    int buff_cycle = BUFF_CYCLE;
    int fix_buff_size = 0;

    // Parse command-line options
    read_line_parameters(argc, argv, rank,
                         &flag_b, &flag_l, &flag_x,
                         &loop_count, &buff_cycle, &fix_buff_size);
    if(flag_x && fix_buff_size >= buff_cycle){buff_cycle = fix_buff_size + 1;}    
    // Print message based on the flags
    if (flag_b && rank == 0) printf("Flag b was set with argument: %d\n", buff_cycle);
    if (flag_l && rank == 0) printf("Flag l was set with argument: %d\n", loop_count);
    if (flag_x && rank == 0) printf("Flag x was set with argument: %d\n", fix_buff_size);

    max_j = (flag_x == 0) ? buff_cycle : (fix_buff_size + 1) ;
    if (rank == 0) printf("buff_cycle: %d loop_count: %d max_j: %d\n", buff_cycle, loop_count, max_j);
    if (flag_x > 0 && rank == 0) printf("fix_buff_size is set as %d\n", fix_buff_size);


    /* -------------------------------------------------------------------------------------------
        NCCL Initialization
    --------------------------------------------------------------------------------------------*/
    ncclUniqueId Id;
    ncclComm_t NCCL_COMM_WORLD, NCCL_COMM_NODE;

    MPI_Barrier(MPI_COMM_WORLD);
    const unsigned long int start_time_nccl_init_us = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();


    ncclGroupStart();
    if (rank == 0) { NCCLCHECK( ncclGetUniqueId(&Id) ); }
    MPI_Bcast(&Id, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
    NCCLCHECK( ncclCommInitRank(&NCCL_COMM_WORLD, size, Id, rank) );
    ncclGroupEnd();

#ifdef PRINT_NCCL_INTRANODE_INFO
    ncclGroupStart();
    if (mynodeid == 0) { NCCLCHECK( ncclGetUniqueId(&Id) ); }
    MPI_Bcast(&Id, sizeof(ncclUniqueId), MPI_BYTE, 0, nodeComm);
    NCCLCHECK( ncclCommInitRank(&NCCL_COMM_NODE, mynodesize, Id, mynodeid) );
    ncclGroupEnd();
    
    int nccl_w_rk;
    int nccl_w_sz;
    ncclGroupStart();
    NCCLCHECK( ncclCommCount(NCCL_COMM_WORLD, &nccl_w_sz)   );
    NCCLCHECK( ncclCommUserRank(NCCL_COMM_WORLD, &nccl_w_rk) );
    ncclGroupEnd();

    int nccl_n_rk;
    int nccl_n_sz;
    ncclGroupStart();
    NCCLCHECK( ncclCommCount(NCCL_COMM_NODE, &nccl_n_sz)   );
    NCCLCHECK( ncclCommUserRank(NCCL_COMM_NODE, &nccl_n_rk) );
    ncclGroupEnd();

    printf("[%d] NCCL_COMM_WORLD: nccl size = %d, nccl rank = %d\n", rank, nccl_w_sz, nccl_w_rk);
    printf("[%d] NCCL_COMM_NODE:  nccl size = %d, nccl rank = %d\n", rank, nccl_n_sz, nccl_n_rk);
    fflush(stdout);
#endif
    MPI_Barrier(MPI_COMM_WORLD);
    const unsigned long int end_time_nccl_init_us = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
    if (rank == 0)
        printf("NCCL init time: %f (s)\n", (end_time_nccl_init_us-start_time_nccl_init_us)/1e6);
    

    // Check if I am one of the destination ranks
    // Read the GPUBENCH_OTOM_DEST environment variable (comma-separated string of ranks)    
    char *dest_ranks_str = getenv("GPUBENCH_OTOM_DEST");
    size_t num_destinations;
    int* dest_ranks = (int*) malloc(size*sizeof(int));
    if (dest_ranks_str != NULL) {
        memset(dest_ranks, 0, size*sizeof(int));
        num_destinations = 0;
        char *token = strtok(dest_ranks_str, ",");
        while (token != NULL) {
            dest_ranks[atoi(token)] = 1;
            ++num_destinations;
	        token = strtok(NULL, ",");
        }
    }else{
        memset(dest_ranks, 1, size*sizeof(int));
        num_destinations = size - 1;
    }


    MPI_Barrier(MPI_COMM_WORLD);


     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    SZTYPE N;
    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    float start_time, stop_time;
    int *error = (int*)malloc(sizeof(int)*buff_cycle);
    int *my_error = (int*)malloc(sizeof(int)*buff_cycle);
    cktype *cpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    cktype *gpu_checks = (cktype*)malloc(sizeof(cktype)*buff_cycle);
    float *elapsed_time = (float*)malloc(sizeof(float)*buff_cycle*loop_count);
    float *inner_elapsed_time = (float*)malloc(sizeof(float)*buff_cycle*loop_count);
    for(int j=fix_buff_size; j<max_j; j++){

        (j!=0) ? (N <<= 1) : (N = 1);
        if (rank == 0) {printf("%i#", j); fflush(stdout);}

        // Allocate memory for A on CPU
        dtype *A;
#ifdef PINNED
        hipHostAlloc(&A, N*sizeof(dtype), hipHostMallocDefault);
#else
        A = (dtype*)malloc(N*sizeof(dtype));
#endif
        cktype *my_cpu_check = (cktype*)malloc(sizeof(cktype));
        cktype *recv_cpu_check = (cktype*)malloc(sizeof(cktype)*size), gpu_check = 0;
        *my_cpu_check = 0U;


        // Initialize all elements of A to 0.0
        for(SZTYPE i=0; i<N; i++) {
            A[i] = 1U * (rank+1);
        }

        dtype *d_A;
        cudaErrorCheck( hipMalloc(&d_A, N*sizeof(dtype)) );
        cudaErrorCheck( hipMemcpy(d_A, A, N*sizeof(dtype), hipMemcpyHostToDevice) );
        gpu_device_reduce_max(d_A, N, my_cpu_check);


        /*

        Implemetantion goes here

        */
        hipEvent_t start, stop;
        cudaErrorCheck(hipEventCreate(&start));
        cudaErrorCheck(hipEventCreate(&stop));

        for(int i=1-(WARM_UP); i<=loop_count; i++){
            MPI_Barrier(MPI_COMM_WORLD);
            cudaErrorCheck(hipEventRecord(start, NULL));

            ncclGroupStart();
            // Assume the root of the otom is rank 0
            if(rank == 0){
                for (int r=1; r<size; r++){
                    if(dest_ranks[r]){
                        ncclSend(d_A, N, ncclDtype, r, NCCL_COMM_WORLD, NULL);
                    }
                }
            }else if(dest_ranks[rank]){
                ncclRecv(d_A, N, ncclDtype, 0, NCCL_COMM_WORLD, NULL);
            }
            ncclGroupEnd();

            cudaErrorCheck(hipEventRecord(stop, NULL));
            cudaErrorCheck(hipEventSynchronize(stop));
            if (i>0) {cudaErrorCheck(hipEventElapsedTime(&(inner_elapsed_time[(j-fix_buff_size)*loop_count+i-1]), start, stop));}

            if (rank == 0) {printf("%%"); fflush(stdout);}
        }
        if (rank == 0) {printf("#\n"); fflush(stdout);}

        gpu_device_reduce(d_A, N, &gpu_check);
        MPI_Allgather(my_cpu_check, 1, MPI_cktype, recv_cpu_check, 1, MPI_cktype, MPI_COMM_WORLD);

        cpu_checks[j] = 0;
        gpu_checks[j] = gpu_check;
        for (int i=0; i<size; i++)
            cpu_checks[j] += recv_cpu_check[i];
        my_error[j] = abs(gpu_checks[j] - cpu_checks[j]);

        cudaErrorCheck( hipFree(d_A) );
        free(recv_cpu_check);
        free(my_cpu_check);
#ifdef PINNED
        hipHostFree(A);
#else
        free(A);
#endif
    }

    if (fix_buff_size<=30) {
        N = 1 << (fix_buff_size - 1);
    } else {
        N = 1 << 30;
        N <<= (fix_buff_size - 31);
    }

    MPI_Allreduce(my_error, error, buff_cycle, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(inner_elapsed_time, elapsed_time, buff_cycle*loop_count, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
    for(int j=fix_buff_size; j<max_j; j++) {
        (j!=0) ? (N <<= 1) : (N = 1);

        SZTYPE num_B, int_num_GB;
        double num_GB;

        num_B = sizeof(dtype)*N*num_destinations;
        // TODO: maybe we can avoid if and just divide always by B_in_GB
        if (j < 31) {
            SZTYPE B_in_GB = 1 << 30;
            num_GB = (double)num_B / (double)B_in_GB;
        } else {
            SZTYPE M = 1 << (j - 30);            
            num_GB = sizeof(dtype)*M*num_destinations;
        }

        double avg_time_per_transfer = 0.0;
        for (int i=0; i<loop_count; i++) {
            elapsed_time[(j-fix_buff_size)*loop_count+i] *= 0.001;
            avg_time_per_transfer += elapsed_time[(j-fix_buff_size)*loop_count+i];
            if(rank == 0) printf("\tTransfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Iteration %d\n", num_B, elapsed_time[(j-fix_buff_size)*loop_count+i], num_GB/elapsed_time[(j-fix_buff_size)*loop_count+i], i);
        }
        avg_time_per_transfer /= ((double)loop_count);

        if(rank == 0) printf("[Average] Transfer size (B): %10" PRIu64 ", Transfer Time (s): %15.9f, Bandwidth (GiB/s): %15.9f, Error: %d\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, error[j] );
        fflush(stdout);
    }

    char *s = (char*)malloc(sizeof(char)*(20*buff_cycle + 100));
    sprintf(s, "[%d] recv_cpu_check = %u", rank, cpu_checks[0]);
    for (int i=fix_buff_size; i<max_j; i++) {
        sprintf(s+strlen(s), " %10d", cpu_checks[i]);
    }
    sprintf(s+strlen(s), " (for Error)\n");
    printf("%s", s);
    fflush(stdout);

    sprintf(s, "[%d] gpu_checks = %u", rank, gpu_checks[0]);
    for (int i=fix_buff_size; i<max_j; i++) {
        sprintf(s+strlen(s), " %10d", gpu_checks[i]);
    }
    sprintf(s+strlen(s), " (for Error)\n");
    printf("%s", s);
    fflush(stdout);

    free(error);
    free(my_error);
    free(cpu_checks);
    free(gpu_checks);
    free(elapsed_time);
    free(inner_elapsed_time);
    free(dest_ranks);
    MPI_Finalize();
    return(0);
}
