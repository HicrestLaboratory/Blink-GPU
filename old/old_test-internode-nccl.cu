#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"

#include <nccl.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#if !defined(OPEN_MPI) || !OPEN_MPI
#error This source code uses an Open MPI-specific extension
#endif

/* Needed for MPIX_Query_cuda_support(), below */
#include "mpi-ext.h"


// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
        exit(0);                                                                            \
    }                                                                                     \
}while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define MPI

static int stringCmp( const void *a, const void *b) {
     return strcmp((const char*)a,(const char*)b);

}

int  assignDeviceToProcess(MPI_Comm *nodeComm, int *nnodes, int *mynodeid)
{
#ifdef MPI
      char     host_name[MPI_MAX_PROCESSOR_NAME];
      char (*host_names)[MPI_MAX_PROCESSOR_NAME];

#else
      char     host_name[20];
#endif
      int myrank;
      int gpu_per_node;
      int n, namelen, color, rank, nprocs;
      size_t bytes;
/*
      if (chkseGPU()<1 && 0) {
        fprintf(stderr, "Invalid GPU Serial number\n");
	exit(EXIT_FAILURE);
      }
*/

#ifdef MPI
      MPI_Comm_rank(MPI_COMM_WORLD, &rank);
      MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
      MPI_Get_processor_name(host_name,&namelen);

      bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
      host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

      strcpy(host_names[rank], host_name);

      for (n=0; n<nprocs; n++)
      {
       MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR,n, MPI_COMM_WORLD);
      }


      qsort(host_names, nprocs, sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

      color = 0;

      for (n=0; n<nprocs; n++)
      {
        if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
        if(strcmp(host_name, host_names[n]) == 0) break;
      }

      MPI_Comm_split(MPI_COMM_WORLD, color, 0, nodeComm);

      MPI_Comm_rank(*nodeComm, &myrank);
      MPI_Comm_size(*nodeComm, &gpu_per_node);

      MPI_Allreduce(&color, nnodes, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
      (*mynodeid) = color;
      (*nnodes) ++;

#else
     //*myrank = 0;
     return 0;
#endif

//      printf ("Assigning device %d  to process on node %s rank %d\n",*myrank,  host_name, rank );
      /* Assign device to MPI process, initialize BLAS and probe device properties */
      //hipSetDevice(*myrank);
      return myrank;
}


int main(int argc, char *argv[])
{
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */




    /* -------------------------------------------------------------------------------------------
        MPI Initialization 
    --------------------------------------------------------------------------------------------*/
    MPI_Init(&argc, &argv);

    int size, nnodes;
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int rank, mynode;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    MPI_Status stat;

//     if(size != 2){
//         if(rank == 0){
//             printf("This program requires exactly 2 MPI ranks, but you are attempting to use %d! Exiting...\n", size);
//         }
//         MPI_Finalize();
//         exit(0);
//     }
    int rank2 = size-1;

    // Map MPI ranks to GPUs
    int num_devices = 0;
    cudaErrorCheck( hipGetDeviceCount(&num_devices) );
//     cudaErrorCheck( hipSetDevice(rank % num_devices) );

    MPI_Comm nodeComm;
    int dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
    hipSetDevice(dev);

    int mynodeid = -1, mynodesize = -1;
    MPI_Comm_rank(nodeComm, &mynodeid);
    MPI_Comm_size(nodeComm, &mynodesize);

    /* -------------------------------------------------------------------------------------------
        NCCL Initialization
    --------------------------------------------------------------------------------------------*/
    ncclUniqueId Id;
    ncclComm_t NCCL_COMM_WORLD, NCCL_COMM_NODE;

    ncclGroupStart();
    if (mynodeid == 0) { NCCLCHECK( ncclGetUniqueId(&Id) ); }
    MPI_Bcast(&Id, sizeof(ncclUniqueId), MPI_BYTE, 0, nodeComm);
    NCCLCHECK( ncclCommInitRank(&NCCL_COMM_NODE, mynodesize, Id, mynodeid) );
    ncclGroupEnd();
    MPI_Barrier(MPI_COMM_WORLD);

    ncclGroupStart();
    if (rank == 0) { NCCLCHECK( ncclGetUniqueId(&Id) ); }
    MPI_Bcast(&Id, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
    NCCLCHECK( ncclCommInitRank(&NCCL_COMM_WORLD, size, Id, rank) );
    ncclGroupEnd();

    int nccl_w_rk;
    int nccl_w_sz;
    ncclGroupStart();
    NCCLCHECK( ncclCommCount(NCCL_COMM_WORLD, &nccl_w_sz)   );
    NCCLCHECK( ncclCommUserRank(NCCL_COMM_WORLD, &nccl_w_rk) );
    ncclGroupEnd();

    int nccl_n_rk;
    int nccl_n_sz;
    ncclGroupStart();
    NCCLCHECK( ncclCommCount(NCCL_COMM_NODE, &nccl_n_sz)   );
    NCCLCHECK( ncclCommUserRank(NCCL_COMM_NODE, &nccl_n_rk) );
    ncclGroupEnd();

    printf("[%d] NCCL_COMM_WORLD: nccl size = %d, nccl rank = %d\n", rank, nccl_w_sz, nccl_w_rk);
    printf("[%d] NCCL_COMM_NODE:  nccl size = %d, nccl rank = %d\n", rank, nccl_n_sz, nccl_n_rk);
    fflush(stdout);

    MPI_Barrier(MPI_COMM_WORLD);

     /* -------------------------------------------------------------------------------------------
        Loop from 8 B to 1 GB
    --------------------------------------------------------------------------------------------*/

    if (rank == 0 || rank == rank2) {
        for(int i=0; i<=27; i++){

            long int N = 1 << i;

            // Allocate memory for A on CPU
            double *A = (double*)malloc(N*sizeof(double));
            double *B = (double*)malloc(N*sizeof(double));
            double my_cpu_check = 1.0, recv_cpu_check;

            // Initialize all elements of A to 0.0
            for(int i=0; i<N; i++){
                A[i] = 1.0 * (rank+1) + i * 0.0001;
                my_cpu_check += A[i];
                B[i] = 0.0;
            }

            double *d_B;
            cudaErrorCheck( hipMalloc(&d_B, N*sizeof(double)) );
            cudaErrorCheck( hipMemcpy(d_B, B, N*sizeof(double), hipMemcpyHostToDevice) );

            double *d_A;
            cudaErrorCheck( hipMalloc(&d_A, N*sizeof(double)) );
            cudaErrorCheck( hipMemcpy(d_A, A, N*sizeof(double), hipMemcpyHostToDevice) );

            int tag1 = 10;
            int tag2 = 20;

            int loop_count = 50;
        double start_time, stop_time, elapsed_time;
            start_time = MPI_Wtime();
    /*

    Implemetantion goes here

    */
            for(int i=1; i<=loop_count; i++){

                ncclGroupStart();
                if(rank == 0){
                    ncclSend(d_A, N, ncclDouble, rank2, NCCL_COMM_WORLD, NULL);
                    ncclRecv(d_B, N, ncclDouble, rank2, NCCL_COMM_WORLD, NULL);
                }
                else if(rank == rank2){
                    ncclSend(d_A, N, ncclDouble, 0, NCCL_COMM_WORLD, NULL);
                    ncclRecv(d_B, N, ncclDouble, 0, NCCL_COMM_WORLD, NULL);
                }
                ncclGroupEnd();
            }
            cudaErrorCheck( hipDeviceSynchronize() );



        stop_time = MPI_Wtime();
            elapsed_time = stop_time - start_time;

            cudaErrorCheck( hipMemcpy(B, d_B, sizeof(double)*N, hipMemcpyDeviceToHost) );
            double gpu_check = 1.0;
            for(int i=0; i<N; i++)
                gpu_check += B[i];
            if(rank == 0){
                MPI_Send(&my_cpu_check,   1, MPI_DOUBLE, rank2, tag1, MPI_COMM_WORLD);
                MPI_Recv(&recv_cpu_check, 1, MPI_DOUBLE, rank2, tag2, MPI_COMM_WORLD, &stat);
            } else if(rank == rank2){
                MPI_Recv(&recv_cpu_check, 1, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &stat);
                MPI_Send(&my_cpu_check,   1, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
            }

            long int num_B = 8*N;
            long int B_in_GB = 1 << 30;
            double num_GB = (double)num_B / (double)B_in_GB;
            double avg_time_per_transfer = elapsed_time / (2.0*(double)loop_count);

            if(rank == 0) printf("Transfer size (B): %10li, Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f, Error: %lf\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer, fabs(gpu_check - recv_cpu_check) );
            fflush(stdout);
            cudaErrorCheck( hipFree(d_A) );
            free(A);
        }
    }
    MPI_Finalize();
    return 0;
}
