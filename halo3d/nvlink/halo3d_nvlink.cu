#include "hip/hip_runtime.h"
// Copyright 2009-2018 Sandia Corporation. Under the terms
// of Contract DE-NA0003525 with Sandia Corporation, the U.S.
// Government retains certain rights in this software.
//
// Copyright (c) 2009-2018, Sandia Corporation
// All rights reserved.
//
// Portions are copyright of other developers:
// See the file CONTRIBUTORS.TXT in the top level directory
// the distribution for more information.
//
// This file is part of the SST software package. For license
// information, see the LICENSE file in the top level directory of the
// distribution.

#include <errno.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../../include/experiment_utils.h"
#include "../../include/debug_utils.h"
#include "../../include/hip/hip_runtime_api.h"

#define BLK_SIZE 256
#define GRD_SIZE 4
#define TID_DIGITS 10000

#define dtype float
#define MPI_dtype MPI_FLOAT

void get_position(const int rank, const int pex, const int pey, const int pez,
                  int* myX, int* myY, int* myZ) {
  const int plane = rank % (pex * pey);
  *myY = plane / pex;
  *myX = (plane % pex) != 0 ? (plane % pex) : 0;
  *myZ = rank / (pex * pey);
}

int convert_position_to_rank(const int pX, const int pY, const int pZ,
                             const int myX, const int myY, const int myZ) {
  // Check if we are out of bounds on the grid
  if ((myX < 0) || (myY < 0) || (myZ < 0) || (myX >= pX) || (myY >= pY) ||
      (myZ >= pZ)) {
    return -1;
  } else {
    return (myZ * (pX * pY)) + (myY * pX) + myX;
  }
}

static int stringCmp( const void *a, const void *b) {
     return strcmp((const char*)a,(const char*)b);

}

int  assignDeviceToProcess(MPI_Comm *nodeComm, int *nnodes, int *mynodeid)
{
      char     host_name[MPI_MAX_PROCESSOR_NAME];
      char (*host_names)[MPI_MAX_PROCESSOR_NAME];

      int myrank;
      int gpu_per_node;
      int n, namelen, color, rank, nprocs;
      size_t bytes;

      MPI_Comm_rank(MPI_COMM_WORLD, &rank);
      MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
      MPI_Get_processor_name(host_name,&namelen);

      bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
      host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

      strcpy(host_names[rank], host_name);

      for (n=0; n<nprocs; n++)
      {
       MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR,n, MPI_COMM_WORLD);
      }


      qsort(host_names, nprocs, sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

      color = 0;

      for (n=0; n<nprocs; n++)
      {
        if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
        if(strcmp(host_name, host_names[n]) == 0) break;
      }

      MPI_Comm_split(MPI_COMM_WORLD, color, 0, nodeComm);

      MPI_Comm_rank(*nodeComm, &myrank);
      MPI_Comm_size(*nodeComm, &gpu_per_node);

      MPI_Allreduce(&color, nnodes, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
      (*mynodeid) = color;
      (*nnodes) ++;

//      printf ("Assigning device %d  to process on node %s rank %d\n",*myrank,  host_name, rank );
      /* Assign device to MPI process, initialize BLAS and probe device properties */
      //hipSetDevice(*myrank);
      return myrank;
}

__global__
void init_kernel(int n, dtype *input, int rank) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  dtype floattid = tid/(dtype)TID_DIGITS;
  dtype val_coord = rank + floattid;
  if (tid < n)
      input[tid] = (dtype)val_coord;

}

__global__
void test_kernel(int n, int ninputs, size_t *sizes, dtype **inputs, dtype *output) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  dtype tmp = 0.0;

  if (tid < n) {
    for (int i=0; i<ninputs; i++)
      if (tid < sizes[i])
        tmp += inputs[i][tid];
  }
  output[tid] = tmp;

}

int main(int argc, char* argv[]) {
  MPI_Init(&argc, &argv);

  int me = -1, mynode = -1;
  int world = -1, nnodes = -1;
  double timeTakenMPI = 0.0, timeTakenCUDA = 0.0, TotalTimeTaken = 0.0;

  MPI_Comm_rank(MPI_COMM_WORLD, &me);
  MPI_Comm_size(MPI_COMM_WORLD, &world);

  int pex = world;
  int pey = 1;
  int pez = 1;

  int nx = 10;
  int ny = 10;
  int nz = 10;

  int repeats = 100;
  int vars = 1;

  long sleep = 1000;

  for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "-nx") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -nx without a value.\n");
        }

        exit(-1);
      }

      nx = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-ny") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -ny without a value.\n");
        }

        exit(-1);
      }

      ny = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-nz") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -nz without a value.\n");
        }

        exit(-1);
      }

      nz = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-pex") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -pex without a value.\n");
        }

        exit(-1);
      }

      pex = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-pey") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -pey without a value.\n");
        }

        exit(-1);
      }

      pey = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-pez") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -pez without a value.\n");
        }

        exit(-1);
      }

      pez = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-iterations") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -iterations without a value.\n");
        }

        exit(-1);
      }

      repeats = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-vars") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -vars without a value.\n");
        }

        exit(-1);
      }

      vars = atoi(argv[i + 1]);
      ++i;
    } else if (strcmp(argv[i], "-sleep") == 0) {
      if (i == argc) {
        if (me == 0) {
          fprintf(stderr, "Error: specified -sleep without a value.\n");
        }

        exit(-1);
      }

      sleep = atol(argv[i + 1]);
      ++i;
    } else {
      if (0 == me) {
        fprintf(stderr, "Unknown option: %s\n", argv[i]);
      }

      exit(-1);
    }
  }

  MPI_Barrier(MPI_COMM_WORLD);

  if ((pex * pey * pez) != world) {
    if (0 == me) {
      fprintf(stderr, "Error: rank grid does not equal number of ranks.\n");
      fprintf(stderr, "%7d x %7d x %7d != %7d\n", pex, pey, pez, world);
    }

    exit(-1);
  }

  MPI_Barrier(MPI_COMM_WORLD);

  if (me == 0) {
    printf("# MPI Nearest Neighbor Communication\n");
    printf("# Info:\n");
    printf("# Processor Grid:         %7d x %7d x %7d\n", pex, pey, pez);
    printf("# Data Grid (per rank):   %7d x %7d x %7d\n", nx, ny, nz);
    printf("# Iterations:             %7d\n", repeats);
    printf("# Variables:              %7d\n", vars);
    printf("# Sleep:                  %7ld\n", sleep);
  }

  // ----------------------------------------------------------------------------------------------
  // PICO asign device
  int dev, deviceCount = 0;

  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (me == 0) {
    printf("#\n");
    printf("# Number of GPUs: %d\n", deviceCount);
  }

  MPI_Comm nodeComm;
  dev = assignDeviceToProcess(&nodeComm, &nnodes, &mynode);
  hipSetDevice(dev);

  int mynodeid = -1, mynodesize = -1;
  MPI_Comm_rank(nodeComm, &mynodeid);
  MPI_Comm_size(nodeComm, &mynodesize);

  MPI_ALL_PRINT( fprintf(fp, "mydev is %d, mynode is %d, nnodes are %d, mynodeid is %d and mynodesize is %d\n", dev, mynode, nnodes, mynodeid, mynodesize); )

  DBG_CHECK(1)

  MPI_Barrier(MPI_COMM_WORLD);
  for (int i=0; i<world; i++) {
    if (me == i)
      printf("#\tMPI process %d has device %d\n", me, dev);
    MPI_Barrier(MPI_COMM_WORLD);
  }
  fflush(stdout);
  MPI_Barrier(MPI_COMM_WORLD);
  // ----------------------------------------------------------------------------------------------

  int posX, posY, posZ;
  get_position(me, pex, pey, pez, &posX, &posY, &posZ);

  int xUp = convert_position_to_rank(pex, pey, pez, posX + 1, posY, posZ);
  int xDown = convert_position_to_rank(pex, pey, pez, posX - 1, posY, posZ);
  int yUp = convert_position_to_rank(pex, pey, pez, posX, posY + 1, posZ);
  int yDown = convert_position_to_rank(pex, pey, pez, posX, posY - 1, posZ);
  int zUp = convert_position_to_rank(pex, pey, pez, posX, posY, posZ + 1);
  int zDown = convert_position_to_rank(pex, pey, pez, posX, posY, posZ - 1);

  size_t xSize = ny * nz * vars, ySize = nx * nz * vars, zSize = nx * ny * vars;

  int requestcount = 0;
  MPI_Status* status;
  status = (MPI_Status*)malloc(sizeof(MPI_Status) * 4 * 2);

  MPI_Request* requests;
  requests = (MPI_Request*)malloc(sizeof(MPI_Request) * 4 * 2);

  // ---------------------------------------
  // PICO enable peer access
  STR_COLL_DEF
  STR_COLL_INIT

  // Pick all the devices that can access each other's memory for this test
  // Keep in mind that CUDA has minimal support for fork() without a
  // corresponding exec() in the child process, but in this case our
  // spawnProcess will always exec, so no need to worry.
  hipDeviceProp_t prop;
  int allPeers = 1, myIPC = 1, allIPC;
  checkCudaErrors(hipGetDeviceProperties(&prop, dev));

  int* canAccesPeer = (int*) malloc(sizeof(int)*deviceCount*deviceCount);
  for (int i = 0; i < deviceCount*deviceCount; i++) canAccesPeer[i] = 0;

  // CUDA IPC is only supported on devices with unified addressing
  if (!prop.unifiedAddressing) {
    STR_COLL_APPEND( sprintf(str_coll.buff, "Device %d does not support unified addressing, skipping...\n", dev); )
    myIPC = 0;
  } else {
    STR_COLL_APPEND( sprintf(str_coll.buff, "Device %d support unified addressing\n", dev); )
  }
  // This sample requires two processes accessing each device, so we need
  // to ensure exclusive or prohibited mode is not set
  if (prop.computeMode != hipComputeModeDefault) {
    STR_COLL_APPEND( sprintf(str_coll.buff, "Device %d is in an unsupported compute mode for this sample\n", dev); )
    myIPC = 0;
  } else {
    STR_COLL_APPEND( sprintf(str_coll.buff, "Device %d is in a supported compute mode for this sample\n", dev); )
  }

  MPI_Allreduce(&myIPC, &allIPC, 1, MPI_INT, MPI_MIN, MPI_COMM_WORLD);
  if (!allIPC) {
    MPI_ALL_PRINT( fprintf(fp, "%s", STR_COLL_GIVE); )
    exit(__LINE__);
  }

  if (me == 0) {
    for (int i = 0; i < deviceCount; i++) {
      for (int j = 0; j < deviceCount; j++) {
        if (j != i) {
          int canAccessPeerIJ, canAccessPeerJI;
          checkCudaErrors( hipDeviceCanAccessPeer(&canAccessPeerJI, j, i) );
          checkCudaErrors( hipDeviceCanAccessPeer(&canAccessPeerIJ, i, j) );

          canAccesPeer[i * deviceCount + j] = (canAccessPeerIJ) ? 1 : 0;
          canAccesPeer[j * deviceCount + i] = (canAccessPeerJI) ? 1 : 0;
          if (!canAccessPeerIJ || !canAccessPeerJI) allPeers = 0;
        } else {
          canAccesPeer[i * deviceCount + j] = -1;
        }
      }
    }
  }

  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Bcast(&allPeers, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(canAccesPeer, deviceCount*deviceCount, MPI_INT, 0, MPI_COMM_WORLD);

  if (allPeers) {
    // Enable peers here.  This isn't necessary for IPC, but it will
    // setup the peers for the device.  For systems that only allow 8
    // peers per GPU at a time, this acts to remove devices from CanAccessPeer
    for (int j = 0; j < deviceCount; j++) {
      if (j != dev) {
        checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
        STR_COLL_APPEND( sprintf(str_coll.buff, "Enabled access from device %d to device %d\n", dev, j); )
      }
    }
  } else {
    if (me == 0) printf(str_coll.buff, "CUDA IPC is not supported by all the node's GPUs\n");
  }

  MPI_ALL_PRINT(
    fprintf(fp, "%s", STR_COLL_GIVE);
    FPRINT_MATRIX(fp, canAccesPeer, deviceCount, deviceCount)
  )
  STR_COLL_FREE
  MPI_Barrier(MPI_COMM_WORLD);
  // ---------------------------------------

  // ---------------------------------------------------------------------------------------
  dtype *dev_xUpSendBuffer, *dev_xUpRecvBuffer, *dev_xDownSendBuffer, *dev_xDownRecvBuffer;
  checkCudaErrors( hipMalloc(&dev_xUpSendBuffer, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMalloc(&dev_xUpRecvBuffer, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMalloc(&dev_xDownSendBuffer, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMalloc(&dev_xDownRecvBuffer, sizeof(dtype) * xSize) );
  // ---------------------------------------------------------------------------------------

  // ---------------------------------------
  checkCudaErrors( hipMemset(dev_xUpSendBuffer, 0, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMemset(dev_xUpRecvBuffer, 0, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMemset(dev_xDownSendBuffer, 0, sizeof(dtype) * xSize) );
  checkCudaErrors( hipMemset(dev_xDownRecvBuffer, 0, sizeof(dtype) * xSize) );

  {
    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(GRD_SIZE, 1, 1);
    init_kernel<<<grid_size, block_size>>>(xSize, dev_xUpSendBuffer, me);
    init_kernel<<<grid_size, block_size>>>(xSize, dev_xDownSendBuffer, me);
    checkCudaErrors( hipDeviceSynchronize() );
  }
  // ---------------------------------------

  // ---------------------------------------------------------------------------------------
  dtype *dev_yUpSendBuffer, *dev_yUpRecvBuffer, *dev_yDownSendBuffer, *dev_yDownRecvBuffer;
  checkCudaErrors( hipMalloc(&dev_yUpSendBuffer, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMalloc(&dev_yUpRecvBuffer, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMalloc(&dev_yDownSendBuffer, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMalloc(&dev_yDownRecvBuffer, sizeof(dtype) * ySize) );
  // --------------------------------------------------------------------------------------

  // ---------------------------------------
  checkCudaErrors( hipMemset(dev_yUpSendBuffer, 0, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMemset(dev_yUpRecvBuffer, 0, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMemset(dev_yDownSendBuffer, 0, sizeof(dtype) * ySize) );
  checkCudaErrors( hipMemset(dev_yDownRecvBuffer, 0, sizeof(dtype) * ySize) );

  {
    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(GRD_SIZE, 1, 1);
    init_kernel<<<grid_size, block_size>>>(ySize, dev_yUpSendBuffer, me);
    init_kernel<<<grid_size, block_size>>>(ySize, dev_yDownSendBuffer, me);
    checkCudaErrors( hipDeviceSynchronize() );
  }
  // ---------------------------------------


  // ---------------------------------------------------------------------------------------
  dtype *dev_zUpSendBuffer, *dev_zUpRecvBuffer, *dev_zDownSendBuffer, *dev_zDownRecvBuffer;
  checkCudaErrors( hipMalloc(&dev_zUpSendBuffer, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMalloc(&dev_zUpRecvBuffer, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMalloc(&dev_zDownSendBuffer, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMalloc(&dev_zDownRecvBuffer, sizeof(dtype) * zSize) );
  // --------------------------------------------------------------------------------------

  // ---------------------------------------
  checkCudaErrors( hipMemset(dev_zUpSendBuffer, 0, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMemset(dev_zUpRecvBuffer, 0, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMemset(dev_zDownSendBuffer, 0, sizeof(dtype) * zSize) );
  checkCudaErrors( hipMemset(dev_zDownRecvBuffer, 0, sizeof(dtype) * zSize) );

  {
    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(GRD_SIZE, 1, 1);
    init_kernel<<<grid_size, block_size>>>(zSize, dev_zUpSendBuffer, me);
    init_kernel<<<grid_size, block_size>>>(zSize, dev_zDownSendBuffer, me);
    checkCudaErrors( hipDeviceSynchronize() );
  }
  // ---------------------------------------

  // ---------------------------------------
  {
    float tmp[3][2], *tmp0;
    srand((unsigned int)time(NULL));
    int x = rand() % (GRD_SIZE*BLK_SIZE);
    int size = (xSize > ySize) ? xSize : ySize;
    if (zSize > size) size = zSize;
    tmp0 = (dtype*)malloc(sizeof(dtype)*(size));
    for (int i=0; i<6; i++) tmp[i/2][i%2] = 0.0;
    checkCudaErrors( hipMemcpy(tmp0, dev_xUpSendBuffer,   xSize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[0][0] = tmp0[x];
    checkCudaErrors( hipMemcpy(tmp0, dev_xDownSendBuffer, xSize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[0][1] = tmp0[x];
    checkCudaErrors( hipMemcpy(tmp0, dev_yUpSendBuffer,   ySize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[1][0] = tmp0[x];
    checkCudaErrors( hipMemcpy(tmp0, dev_yDownSendBuffer, ySize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[1][1] = tmp0[x];
    checkCudaErrors( hipMemcpy(tmp0, dev_zUpSendBuffer,   zSize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[2][0] = tmp0[x];
    checkCudaErrors( hipMemcpy(tmp0, dev_zDownSendBuffer, zSize*sizeof(float), hipMemcpyDeviceToHost) );
    tmp[2][1] = tmp0[x];
    checkCudaErrors( hipDeviceSynchronize() );

    MPI_ALL_PRINT(
      fprintf(fp, "extracted tid = %d\n", x);
      fprintf(fp, "xUpSendBuffer = %6.4f, xDownSendBuffer = %6.4f\n", tmp[0][0], tmp[0][1]);
      fprintf(fp, "yUpSendBuffer = %6.4f, yDownSendBuffer = %6.4f\n", tmp[1][0], tmp[1][1]);
      fprintf(fp, "zUpSendBuffer = %6.4f, zDownSendBuffer = %6.4f\n", tmp[2][0], tmp[2][1]);
    )
    free(tmp0);
  }
  MPI_Barrier(MPI_COMM_WORLD);
  // ---------------------------------------

  struct timeval start;
  struct timeval end;

  struct timespec sleepTS;
  sleepTS.tv_sec = 0;
  sleepTS.tv_nsec = sleep;

  struct timespec remainTS;

  INIT_EXPS
  TIMER_DEF(0);
  SET_EXPERIMENT_NAME(0, "halo3d")
  SET_EXPERIMENT_TYPE(0, "nvlink")
  SET_EXPERIMENT(0, "CUDA")

  SET_EXPERIMENT_NAME(1, "halo3d")
  SET_EXPERIMENT_TYPE(1, "nvlink")
  SET_EXPERIMENT(1, "MPI")

  SET_EXPERIMENT_NAME(2, "halo3d")
  SET_EXPERIMENT_TYPE(2, "nvlink")
  SET_EXPERIMENT(2, "TOTAL")

  if (nnodes > 1) {
    SET_EXPERIMENT_LAYOUT(0, "interNodes")
    SET_EXPERIMENT_LAYOUT(1, "interNodes")
    SET_EXPERIMENT_LAYOUT(2, "interNodes")
  } else {
    SET_EXPERIMENT_LAYOUT(0, "intraNode")
    SET_EXPERIMENT_LAYOUT(1, "intraNode")
    SET_EXPERIMENT_LAYOUT(2, "intraNode")
  }


  if (nnodes != 1) {
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);
    if (me == 0) printf("The NVLINK version is only implemented for intraNode communication\n");
    if (me == 0) PRINT_EXPARIMENT_STATS
    MPI_Barrier(MPI_COMM_WORLD);
    exit(__LINE__);
  }

  dtype *xUpPeerBuffer, *xDownPeerBuffer;
  hipEvent_t xUpSendEvent, xDownSendEvent, xUpRecvEvent, xDownRecvEvent;
  hipIpcMemHandle_t xUpSendHandle, xUpRecvHandle, xDownSendHandle, xDownRecvHandle;
  hipIpcEventHandle_t xUpSendEventHandle, xUpRecvEventHandle, xDownSendEventHandle, xDownRecvEventHandle;

  dtype *yUpPeerBuffer, *yDownPeerBuffer;
  hipEvent_t yUpSendEvent, yDownSendEvent, yUpRecvEvent, yDownRecvEvent;
  hipIpcMemHandle_t yUpSendHandle, yUpRecvHandle, yDownSendHandle, yDownRecvHandle;
  hipIpcEventHandle_t yUpSendEventHandle, yUpRecvEventHandle, yDownSendEventHandle, yDownRecvEventHandle;

  dtype *zUpPeerBuffer, *zDownPeerBuffer;
  hipEvent_t zUpSendEvent, zDownSendEvent, zUpRecvEvent, zDownRecvEvent;
  hipIpcMemHandle_t zUpSendHandle, zUpRecvHandle, zDownSendHandle, zDownRecvHandle;
  hipIpcEventHandle_t zUpSendEventHandle, zUpRecvEventHandle, zDownSendEventHandle, zDownRecvEventHandle;

  gettimeofday(&start, NULL);
  for (int i = 0; i < repeats; ++i) {
    requestcount = 0;

    if (nanosleep(&sleepTS, &remainTS) == EINTR) {
      while (nanosleep(&remainTS, &remainTS) == EINTR)
        ;
    }

    // =================================================================================================================


    // ---------------------------------------
    TIMER_START(0);
    if (xUp > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&xUpSendHandle, dev_xUpSendBuffer) );
      checkCudaErrors( hipEventCreate(&xUpSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&xUpSendEventHandle, xUpSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    if (xDown > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&xDownSendHandle, dev_xDownSendBuffer) );
      checkCudaErrors( hipEventCreate(&xDownSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&xDownSendEventHandle, xDownSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    TIMER_START(0);
    if (xUp > -1) {
      MPI_Irecv(&xUpRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, xUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&xUpRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, xUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&xUpSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, xUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&xUpSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, xUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    if (xDown > -1) {
      MPI_Irecv(&xDownRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, xDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&xDownRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, xDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&xDownSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, xDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&xDownSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, xDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    MPI_Waitall(requestcount, requests, status);
    requestcount = 0;
    TIMER_STOP(0);
    timeTakenMPI += TIMER_ELAPSED(0);

    // ---------------------------------------
    TIMER_START(0);
    if (xUp > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&xUpPeerBuffer, *(hipIpcMemHandle_t*)&xUpRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&xUpRecvEvent, *(hipIpcEventHandle_t *)&xUpRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_xUpRecvBuffer, xUpPeerBuffer, sizeof(dtype)*xSize, hipMemcpyDeviceToDevice) );
    }
    if (xDown > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&xDownPeerBuffer, *(hipIpcMemHandle_t*)&xDownRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&xDownRecvEvent, *(hipIpcEventHandle_t *)&xDownRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_xDownRecvBuffer, xDownPeerBuffer, sizeof(dtype)*xSize, hipMemcpyDeviceToDevice) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // ---------------------------------------
    TIMER_START(0);
    if (xUp > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(xUpPeerBuffer) );
      checkCudaErrors( hipEventDestroy(xUpRecvEvent) );
    }
    if (xDown > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(xDownPeerBuffer) );
      checkCudaErrors( hipEventDestroy(xDownRecvEvent) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // =================================================================================================================

    // ---------------------------------------
    TIMER_START(0);
    if (yUp > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&yUpSendHandle, dev_yUpSendBuffer) );
      checkCudaErrors( hipEventCreate(&yUpSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&yUpSendEventHandle, yUpSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    if (yDown > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&yDownSendHandle, dev_yDownSendBuffer) );
      checkCudaErrors( hipEventCreate(&yDownSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&yDownSendEventHandle, yDownSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    TIMER_START(0);
    if (yUp > -1) {
      MPI_Irecv(&yUpRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, yUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&yUpRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, yUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&yUpSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, yUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&yUpSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, yUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    if (yDown > -1) {
      MPI_Irecv(&yDownRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, yDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&yDownRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, yDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&yDownSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, yDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&yDownSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, yDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    MPI_Waitall(requestcount, requests, status);
    requestcount = 0;
    TIMER_STOP(0);
    timeTakenMPI += TIMER_ELAPSED(0);

    // ---------------------------------------
    TIMER_START(0);
    if (yUp > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&yUpPeerBuffer, *(hipIpcMemHandle_t*)&yUpRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&yUpRecvEvent, *(hipIpcEventHandle_t *)&yUpRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_yUpRecvBuffer, yUpPeerBuffer, sizeof(dtype)*ySize, hipMemcpyDeviceToDevice) );
    }
    if (yDown > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&yDownPeerBuffer, *(hipIpcMemHandle_t*)&yDownRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&yDownRecvEvent, *(hipIpcEventHandle_t *)&yDownRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_yDownRecvBuffer, yDownPeerBuffer, sizeof(dtype)*ySize, hipMemcpyDeviceToDevice) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // ---------------------------------------
    TIMER_START(0);
    if (yUp > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(yUpPeerBuffer) );
      checkCudaErrors( hipEventDestroy(yUpRecvEvent) );
    }
    if (yDown > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(yDownPeerBuffer) );
      checkCudaErrors( hipEventDestroy(yDownRecvEvent) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // =================================================================================================================

    // ---------------------------------------
    TIMER_START(0);
    if (zUp > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&zUpSendHandle, dev_zUpSendBuffer) );
      checkCudaErrors( hipEventCreate(&zUpSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&zUpSendEventHandle, zUpSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    if (zDown > -1) {
      checkCudaErrors( hipIpcGetMemHandle((hipIpcMemHandle_t*)&zDownSendHandle, dev_zDownSendBuffer) );
      checkCudaErrors( hipEventCreate(&zDownSendEvent, hipEventDisableTiming | hipEventInterprocess) );
      checkCudaErrors( hipIpcGetEventHandle((hipIpcEventHandle_t*)&zDownSendEventHandle, zDownSendEvent) );
    }
    MPI_Barrier(MPI_COMM_WORLD);
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    TIMER_START(0);
    if (zUp > -1) {
      MPI_Irecv(&zUpRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, zUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&zUpRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, zUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&zUpSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, zUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&zUpSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, zUp, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    if (zDown > -1) {
      MPI_Irecv(&zDownRecvHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, zDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Irecv(&zDownRecvEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, zDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&zDownSendHandle, sizeof(hipIpcMemHandle_t), MPI_BYTE, zDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
      MPI_Isend(&zDownSendEventHandle, sizeof(hipIpcEventHandle_t), MPI_BYTE, zDown, 1000, MPI_COMM_WORLD, &requests[requestcount++]);
    }

    MPI_Waitall(requestcount, requests, status);
    requestcount = 0;
    TIMER_STOP(0);
    timeTakenMPI += TIMER_ELAPSED(0);

    // ---------------------------------------
    TIMER_START(0);
    if (zUp > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&zUpPeerBuffer, *(hipIpcMemHandle_t*)&zUpRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&zUpRecvEvent, *(hipIpcEventHandle_t *)&zUpRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_zUpRecvBuffer, zUpPeerBuffer, sizeof(dtype)*zSize, hipMemcpyDeviceToDevice) );
    }
    if (zDown > -1) {
      checkCudaErrors( hipIpcOpenMemHandle((void**)&zDownPeerBuffer, *(hipIpcMemHandle_t*)&zDownRecvHandle, hipIpcMemLazyEnablePeerAccess) );
      checkCudaErrors( hipIpcOpenEventHandle(&zDownRecvEvent, *(hipIpcEventHandle_t *)&zDownRecvEventHandle) );

      checkCudaErrors( hipMemcpy(dev_zDownRecvBuffer, zDownPeerBuffer, sizeof(dtype)*zSize, hipMemcpyDeviceToDevice) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // ---------------------------------------
    TIMER_START(0);
    if (zUp > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(zUpPeerBuffer) );
      checkCudaErrors( hipEventDestroy(zUpRecvEvent) );
    }
    if (zDown > -1) {
      checkCudaErrors( hipIpcCloseMemHandle(zDownPeerBuffer) );
      checkCudaErrors( hipEventDestroy(zDownRecvEvent) );
    }
    TIMER_STOP(0);
    timeTakenCUDA += TIMER_ELAPSED(0);
    // ---------------------------------------

    // =================================================================================================================
  }

  gettimeofday(&end, NULL);
  ADD_TIME_EXPERIMENT(0, timeTakenCUDA)
  ADD_TIME_EXPERIMENT(1, timeTakenMPI)
  TotalTimeTaken = timeTakenCUDA + timeTakenMPI;
  ADD_TIME_EXPERIMENT(2, TotalTimeTaken)

  // ---------------------------------------
  // PICO disable peer access
  MPI_Barrier(MPI_COMM_WORLD);
  for (int j = 0; j < deviceCount; j++) {
    if (j != dev) {
      checkCudaErrors(hipDeviceDisablePeerAccess(j));
      printf("[%d] Disable access from device %d to device %d\n", me, dev, j);
    }
  }
  // ---------------------------------------

  MPI_Barrier(MPI_COMM_WORLD);

  // ---------------------------------------
  {
    size_t test_sizes[6], *dev_test_sizes;
    dtype *test_vector[6], **dev_test_vector;
    srand((unsigned int)time(NULL));
    int x = rand() % (GRD_SIZE*BLK_SIZE);
    int maxSize = (xSize > ySize) ? xSize : ySize;
    if (zSize > maxSize) maxSize = zSize;

    dtype *dev_checkVector, *checkVector;
    checkVector = (dtype*) malloc(sizeof(dtype)*maxSize);
    checkCudaErrors( hipMalloc(&dev_checkVector,   sizeof(dtype) * maxSize) );
    checkCudaErrors( hipMemset(dev_checkVector, 0, sizeof(dtype) * maxSize) );

    test_sizes[0] = xSize;
    test_sizes[1] = xSize;
    test_sizes[2] = ySize;
    test_sizes[3] = ySize;
    test_sizes[4] = zSize;
    test_sizes[5] = zSize;
    test_vector[0] = dev_xUpRecvBuffer;
    test_vector[1] = dev_xDownRecvBuffer;
    test_vector[2] = dev_yUpRecvBuffer;
    test_vector[3] = dev_yDownRecvBuffer;
    test_vector[4] = dev_zUpRecvBuffer;
    test_vector[5] = dev_zDownRecvBuffer;

    checkCudaErrors( hipMalloc(&dev_test_sizes,  sizeof(size_t) * 6) );
    checkCudaErrors( hipMalloc(&dev_test_vector, sizeof(dtype*) * 6) );
    checkCudaErrors( hipMemcpy(dev_test_sizes,  test_sizes,  sizeof(size_t) * 6, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(dev_test_vector, test_vector, sizeof(dtype*) * 6, hipMemcpyHostToDevice) );

    {
      dim3 block_size(BLK_SIZE, 1, 1);
      dim3 grid_size(GRD_SIZE, 1, 1);
      test_kernel<<<grid_size, block_size>>>(maxSize, 6, dev_test_sizes, dev_test_vector, dev_checkVector);
      checkCudaErrors( hipDeviceSynchronize() );
    }
    checkCudaErrors( hipMemcpy(checkVector, dev_checkVector, maxSize*sizeof(dtype), hipMemcpyDeviceToHost) );
    checkCudaErrors( hipDeviceSynchronize() );

    MPI_ALL_PRINT(
      fprintf(fp, "xUp = %d, xDown = %d, yUp = %d, yDown = %d, zUp = %d, zDown = %d\n", xUp, xDown, yUp, yDown, zUp, zDown);
      fprintf(fp, "extracted tid = %d\n", x);
      fprintf(fp, "checkVector = %6.4f\n", checkVector[x]);
    )

    checkCudaErrors( hipFree(dev_test_vector) );
    checkCudaErrors( hipFree(dev_checkVector) );
    checkCudaErrors( hipFree(dev_test_sizes) );
    free(checkVector);
  }
  MPI_Barrier(MPI_COMM_WORLD);
  // ---------------------------------------

  if (convert_position_to_rank(pex, pey, pez, pex / 2, pey / 2, pez / 2) == me) {
    printf("# Results from rank: %d\n", me);

    const double timeTaken =
        (((double)end.tv_sec) + ((double)end.tv_usec) * 1.0e-6) -
        (((double)start.tv_sec) + ((double)start.tv_usec) * 1.0e-6);
    const double bytesXchng =
        ((double)(xUp > -1 ? sizeof(dtype) * xSize * 2 : 0)) +
        ((double)(xDown > -1 ? sizeof(dtype) * xSize * 2 : 0)) +
        ((double)(yUp > -1 ? sizeof(dtype) * ySize * 2 : 0)) +
        ((double)(yDown > -1 ? sizeof(dtype) * ySize * 2 : 0)) +
        ((double)(zUp > -1 ? sizeof(dtype) * zSize * 2 : 0)) +
        ((double)(zDown > -1 ? sizeof(dtype) * zSize * 2 : 0));

    printf("# %20s %20s %20s\n", "Time", "KBytesXchng/Rank-Max", "MB/S/Rank");
    printf("  %20.6f %20.4f %20.4f\n", timeTaken, bytesXchng / 1024.0,
           (bytesXchng / 1024.0) / timeTaken);
  }

  fflush(stdout);
  MPI_Barrier(MPI_COMM_WORLD);
  if (me == 0) PRINT_EXPARIMENT_STATS

  MPI_Finalize();
}
